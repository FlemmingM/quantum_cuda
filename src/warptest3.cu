#include <hip/hip_runtime.h>
#include <iostream>
#include <omp.h>


// Kernel for GPU computations
__global__ void kernel(float* data, long int offset, int device_id) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < offset) {
        // printf("hello from device %d, idx %d\n", device_id, idx);
        data[idx] = idx * 1.0;
    }
}

// Error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error in " << __FILE__ << "@" << __LINE__ << ": " \
                      << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)


int main() {

    #define N 16  // Size of the array
    const int num_threads = 4;

    int chunks[num_threads];

    for (int i = 0; i < num_threads; ++i) {
        chunks[i] = N / num_threads;
    }

    dim3 dimBlock(256);
    dim3 dimGrid((N/num_threads + dimBlock.x - 1) / dimBlock.x);


    float *d_data[num_threads];
    float *h_data[num_threads];

    hipStream_t streams[num_threads];


    double time = omp_get_wtime();

    #pragma omp parallel for num_threads(num_threads)
    for (int i = 0; i < num_threads; ++i) {
        // Create streams for parallel execution

        CUDA_CHECK(hipStreamCreate(&streams[i]));
        CUDA_CHECK(hipHostMalloc((void**)&h_data[i], chunks[i] * sizeof(float), hipHostMallocDefault));
        CUDA_CHECK(hipMalloc((void**)&d_data[i], chunks[i] * sizeof(float)));

        double time2 = omp_get_wtime();
        kernel<<<dimGrid, dimBlock, 0, streams[i]>>>(d_data[i], chunks[i], 0);
        double elapsed2 = omp_get_wtime() - time2;

        CUDA_CHECK(hipMemcpyAsync(h_data[i], d_data[i], chunks[i] * sizeof(float), hipMemcpyDeviceToHost, streams[i]));

        // Synchronize streams
        CUDA_CHECK(hipStreamSynchronize(streams[i]));

        // Destroy the stream
        CUDA_CHECK(hipStreamDestroy(streams[i]));
}

    double elapsed = omp_get_wtime() - time;
    printf("Time: %f \n", elapsed);

    for (int i = 0; i < num_threads; ++i) {
        printf("[ ");
        for (int j=0; j < chunks[i]; j++) {
            printf("%f ", h_data[i][j]);
        }
        printf("]\n");
    }


    // Cleanup
    for (int i = 0; i < num_threads; ++i) {
        CUDA_CHECK(hipFree(d_data[i]));
        CUDA_CHECK(hipHostFree(h_data[i]));
    }

    return 0;
}
