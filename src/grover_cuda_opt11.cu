#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <math.h>
#include <string.h>
#include <errno.h>
#include <omp.h>
#include "utils_cuda_opt11.h"

typedef hipDoubleComplex Complex;

int main(int argc, char* argv[]) {

    // collect input args
    // if (argc < 6) {
    //     fprintf(stderr, "Usage: %s n qubits<int>; marked state<int>; number of samples<int>; fileName<string>; verbose 0 or 1<int>\n", argv[0]);
    //     return 1;
    // }

    int n = atoi(argv[1]);
    long long int N = (long long int)pow(2, n);
    long long int markedState = atoi(argv[2]);
    int block_size = atoi(argv[3]);
    // const char* fileName = argv[4];
    // int verbose = atoi(argv[5]);

    if (markedState > (N-1)) {
        fprintf(stderr, "You chose a markedState %d but the largest state possible is state %d", markedState, (N-1));
        return 1;
    }

    // Define the gates
    hipDoubleComplex H_h[4] = {
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0),
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(-1.0 / sqrt(2.0), 0.0)
    };

    hipDoubleComplex X_H_h[4] = {
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(-1.0 / sqrt(2.0), 0.0),
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0)
    };

    hipDoubleComplex I_h[4] = {
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0),
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(1.0, 0.0)
    };

    hipDoubleComplex Z_h[4] = {
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0),
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(-1.0, 0.0)
    };

    hipDoubleComplex X_h[4] = {
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(1.0, 0.0),
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0)
    };

    Complex *state_h;
    Complex *state_d;
    // Complex *new_state_h;
    // Complex *new_state_d;
    Complex *H_d;
    Complex *X_H_d;
    Complex *I_d;
    Complex *Z_d;
    Complex *X_d;


    int *new_idx_d;
    int *old_idx_d;
    int *old_linear_idxs_h;
    int *old_linear_idxs_d;

    // Malloc on device and host
    // Init the state
    hipHostMalloc((void **)&state_h, N * sizeof(Complex));
    hipMalloc((void **)&state_d, N * sizeof(Complex));
    // Init the |0>^(xn) state and the new_state
    // state_h[0] = make_hipDoubleComplex(1.0, 0.0);
    // for (int i = 1; i < N; ++i) {
    //     state_h[i] = make_hipDoubleComplex(0.0, 0.0);
    // }
    // hipMemcpy(state_d, state_h, N * sizeof(Complex), hipMemcpyHostToDevice);


    // Malloc the gate on device
    hipMalloc((void **)&H_d, 4 * sizeof(Complex));
    hipMalloc((void **)&X_H_d, 4 * sizeof(Complex));
    hipMalloc((void **)&I_d, 4 * sizeof(Complex));
    hipMalloc((void **)&Z_d, 4 * sizeof(Complex));
    hipMalloc((void **)&X_d, 4 * sizeof(Complex));

    // Copy from host to device
    hipMemcpy(H_d, H_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
    hipMemcpy(X_H_d, X_H_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
    hipMemcpy(I_d, I_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
    hipMemcpy(Z_d, Z_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
    hipMemcpy(X_d, X_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);

    dim3 dimBlock(block_size);
    dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x);

    // const int blockSize = val;
    // const int gridSize = (N + blockSize - 1) / blockSize;

    // Allocate shared memory for reduction
    // int sharedMemSize = blockSize * sizeof(Complex);
    int sharedMemSize = 2*N * sizeof(Complex);
    int sharedMemSize2 = 2*N * sizeof(int);

    // Malloc the indices on the device
    hipMalloc(&new_idx_d, dimGrid.x * dimBlock.x * n * sizeof(int));
    hipMalloc(&old_idx_d, dimGrid.x * dimBlock.x * n * sizeof(int));

    // hipHostMalloc(&old_linear_idxs_h, 2 * N * n * sizeof(int));
    hipMalloc(&old_linear_idxs_d, 2 * N * n * sizeof(int));
    // hipMalloc(&old_linear_idxs_d, gridSize * blockSize * 2 * n * sizeof(int));
    // hipMalloc(&old_linear_idxs_d, gridSize * blockSize * 2 * n * sizeof(int));

    // Assuming we have t = 1 solution in grover's algorithm
    // we have k = floor(pi/4 * sqrt(N))
    int k = (int)floor(M_PI / 4 * sqrt(N));



    double time = omp_get_wtime();

    zeroOutState<<<dimGrid, dimBlock>>>(state_d, N);

    for (int i = 0; i < n; ++i) {
        compute_idx<<<dimGrid, dimBlock, sharedMemSize2>>>(i, new_idx_d, old_idx_d, n, N, old_linear_idxs_d);
    }
    // hipMemcpy(old_linear_idxs_h, old_linear_idxs_d, 2*N* n * sizeof(int), hipMemcpyDeviceToHost);


    // for (int i = 0; i < (2*N*n); ++i) {
    //     printf("%d ", old_linear_idxs_h[i]);
    // }
    // contract_tensor<<<gridSize, blockSize, sharedMemSize>>>(state_d, H_d, 0, new_idx_d, old_idx_d, n, N, old_linear_idxs_d);
    // contract_tensor<<<gridSize, blockSize>>>(state_d, H_d, 0, new_state_d, shape_d, new_idx_d, old_idx_d, n, N);

        // contract_tensor_baseline<<<dimGrid, dimBlock>>>(state, gate, i, new_state, shape, n, N);
        // hipDeviceSynchronize();
        // Update the state with the new state
    // updateState<<<gridSize, blockSize>>>(state_d, new_state_d, N);





    // Now apply the H gate n times, once for each qubit
    applyGateAllQubits(state_d, H_d, new_idx_d, old_idx_d, n, N, dimBlock, dimGrid, sharedMemSize, old_linear_idxs_d);

    // hipDeviceSynchronize();


    // Apply Grover's algorithm k iteration and then sample
    // if (verbose == 1) {
    //     printf("Running %d round(s)\n", k);
    // }

    for (int i = 0; i < k; ++i) {
        applyPhaseFlip<<<dimGrid, dimBlock>>>(state_d, markedState);
        applyDiffusionOperator(state_d, X_H_d, H_d, X_d, Z_d, new_idx_d, old_idx_d, n, N, dimBlock, dimGrid, sharedMemSize, old_linear_idxs_d);
        // hipDeviceSynchronize();
    }

    hipDeviceSynchronize();
    double elapsed = omp_get_wtime() - time;
    printf("Time: %f \n", elapsed);


    hipMemcpy(state_h, state_d, N * sizeof(Complex), hipMemcpyDeviceToHost);

    // if (verbose == 1) {
    // printState(state_h, N, "Initial state");
    // }

    // // Apply Grover's algorithm k iteration and then sample
    // if (verbose == 1) {
    //     printf("Running %d round(s)\n", k);
    // }

    // double time = omp_get_wtime();

    // for (int i = 0; i < k; ++i) {
    //     if (verbose == 1) {
    //         printf("%d/%d\n", i, k);
    //     }
    //     // Apply Oracle
    //     applyPhaseFlip(state, markedState);
    //     if (verbose == 1) {
    //         printState(state, N, "Oracle applied");
    //     }
    //     // Apply the diffusion operator
    //     applyDiffusionOperator(state, new_state, shape, H, X, Z, n, N);
    //     if (verbose == 1) {
    //         printState(state, N, "After Diffusion");
    //     }
    // }

    // double elapsed = omp_get_wtime() - time;
    // printf("Time: %f \n", elapsed);

    // // Sample the states wheighted by their amplitudes
    // double* averages = simulate(state_h, N, 1);
    // if (verbose == 1) {
    //     printf("Average frequency per position:\n");
    //     for (int i = 0; i < N; ++i) {
    //         printf("Position %d: %f\n", i, averages[i]);
    //     }
    // }


    // // save the data
    // saveArrayToCSV(averages, N, 'test.csv');

    hipFree(state_d);
    hipFree(H_d);
    hipFree(old_linear_idxs_d);
    hipFree(I_d);
    hipFree(Z_d);
    hipFree(X_d);
    hipHostFree(state_h);
    hipHostFree(H_h);
    hipHostFree(I_h);
    hipHostFree(Z_h);
    hipHostFree(X_h);

    return 0;
}