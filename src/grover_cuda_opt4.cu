#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <math.h>
#include <string.h>
#include <errno.h>
#include <omp.h>
#include "utils_cuda_opt4.h"

typedef hipDoubleComplex Complex;

int main(int argc, char* argv[]) {

    // collect input args
    // if (argc < 6) {
    //     fprintf(stderr, "Usage: %s n qubits<int>; marked state<int>; number of samples<int>; fileName<string>; verbose 0 or 1<int>\n", argv[0]);
    //     return 1;
    // }

    int n = atoi(argv[1]);
    long long int N = (long long int)pow(2, n);
    long long int markedState = atoi(argv[2]);
    const int num_chunks_per_group = atoi(argv[3]);
    const int block_size = atoi(argv[4]);
    // const char* fileName = argv[4];
    // int verbose = atoi(argv[5]);

    if (markedState > (N-1)) {
        fprintf(stderr, "You chose a markedState %d but the largest state possible is state %d", markedState, (N-1));
        return 1;
    }


    // Define the number of groups to do the parallel search with more than 10 qubits
    // while still using the fast shared memory

    long long int num_groups = pow(2, ((n - 10 < 0) ? 0 : (n - 10)));
    printf("num_groups: %d\n", num_groups);
    // printf("test: %d\n", log2)

    long long int num_chunks = num_groups * num_chunks_per_group;
    int qubits_per_chunk = (int)(n-log2((double)num_chunks));
    printf("num chunks: %d, n per chunk: %d\n",num_chunks, qubits_per_chunk);

    // Define the config for threads, devices and streams
    // const int num_chunks = 2;

    int chunks[num_chunks];
    // int sharedMemSizes[num_chunks];
    int sharedMemSize = (int)(pow(2, 11)) * sizeof(Complex);
    printf("Using shared memory size: %d, per group[%d]: %d\n", sharedMemSize, num_chunks_per_group, sharedMemSize/num_chunks_per_group);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    if (sharedMemSize > deviceProp.sharedMemPerBlock) {
        printf("Requested shared memory size (%d bytes) exceeds the device limit (%d bytes).\n", sharedMemSize, deviceProp.sharedMemPerBlock);
        return -1;
    }

    for (int i = 0; i < num_chunks; ++i) {
        chunks[i] = N / num_chunks;
    }


    for (int i = 0; i < num_chunks; ++i) {
        chunks[i] = N / num_chunks;
    }
    printf("Using chunk size for computation: %d\n", chunks[0]);

    // Define the chunk for the oracle
    long long int oracle_chunk = markedState / (N / num_chunks);
    markedState = markedState % (N / num_chunks);
    long long int recoveredState = oracle_chunk*(N / num_chunks)+markedState;
    printf("oracle_chunk: %lld, pos: %lld, recovered: %lld\n", oracle_chunk, markedState, recoveredState);

    // dim3 dimBlock(block_size);
    // dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x);

    dim3 dimBlock((int)(pow(2, qubits_per_chunk)));
    dim3 dimGrid(num_chunks_per_group);

    // Set the gates:
    int num_devices = 1;
    Complex *H_d[num_devices];
    Complex *I_d[num_devices];
    Complex *Z_d[num_devices];
    Complex *X_d[num_devices];
    Complex *X_H_d[num_devices];
    allocateGatesDevice(num_devices, H_d, I_d, Z_d, X_d, X_H_d);

    // Init the arrays
    Complex *state_h[num_chunks];
    Complex *state_d[num_chunks];
    int *new_idx_d[num_chunks];
    int *old_idx_d[num_chunks];

    for (int i = 0; i < num_chunks; ++i) {
         // Init the state
        hipHostMalloc((void **)&state_h[i], chunks[i] * sizeof(Complex));
        // hipMalloc((void **)&state_d[i], chunks[i] * sizeof(Complex));
        // Init the |0>^(xn) state and the new_state
        state_h[i][0] = make_hipDoubleComplex(1.0, 0.0);
        for (int j = 1; j < chunks[i]; ++j) {
            state_h[i][j] = make_hipDoubleComplex(0.0, 0.0);
        }
        // hipMemcpy(state_d[i], state_h[i], chunks[i] * sizeof(Complex), hipMemcpyHostToDevice);

        // Malloc the indices on the device
        // hipMalloc(&new_idx_d[i], dimGrid.x * dimBlock.x * qubits_per_chunk * sizeof(int));
        // hipMalloc(&old_idx_d[i], dimGrid.x * dimBlock.x * qubits_per_chunk * sizeof(int));
        hipMalloc(&new_idx_d[i], (int)(pow(2, qubits_per_chunk)) * qubits_per_chunk * sizeof(int));
        hipMalloc(&old_idx_d[i], (int)(pow(2, qubits_per_chunk)) * qubits_per_chunk * sizeof(int));
    }






    // // Assuming we have t = 1 solution in grover's algorithm
    // // we have k = floor(pi/4 * sqrt(N/num_chunks))
    int k = (int)floor(M_PI / 4 * sqrt(N/num_chunks));
    printf("running %d rounds\n", k);


    double time = omp_get_wtime();

    hipStream_t streams[num_chunks];
    for (int i = 0; i < num_chunks; ++i) {
        hipStreamCreate(&streams[i]);
        // hipMalloc((void **)&state_d[j*num_chunks_per_group+i], chunks[j*num_chunks_per_group+i] * sizeof(Complex));
    }

    // #pragma omp parallel for collapse(2) num_threads(num_groups * num_chunks_per_group)
    // for (int j = 0; j < num_groups; ++j) {
    //     for (int i = 0; i < num_chunks_per_group; ++i) {
    //         hipMalloc((void **)&state_d[j*num_chunks_per_group+i], chunks[j*num_chunks_per_group+i] * sizeof(Complex));
    //     }
    // }

    for (int j = 0; j < num_groups; ++j) {
        // #pragma omp parallel for num_threads(num_groups)
        for (int i = 0; i < num_chunks_per_group; ++i) {
            // hipStreamCreate(&streams[i]);
            // tid = omp_get_thread_num();
            // printf("Welcome to GFG from thread = %d\n", tid);

            // printf("j %d, i %d, num_chunks_per_group %d, j*num_chunks_per_group+i %d\n", j, i, num_chunks_per_group, j*num_chunks_per_group+i);
            hipMalloc((void **)&state_d[j*num_chunks_per_group+i], chunks[j*num_chunks_per_group+i] * sizeof(Complex));
            hipMemcpyAsync(state_d[j*num_chunks_per_group+i], state_h[j*num_chunks_per_group+i], chunks[j*num_chunks_per_group+i] * sizeof(Complex), hipMemcpyHostToDevice, streams[j*num_chunks_per_group+i]);
            // contract_tensor<<<num_chunks_per_group, (int)(pow(2, qubits_per_chunk)) , sharedMemSize, streams[j*num_chunks_per_group+i]>>>(state_d[j*num_chunks_per_group+i], H_d[0], 0, new_idx_d[j*num_chunks_per_group+i], old_idx_d[j*num_chunks_per_group+i], qubits_per_chunk, i*chunks[j*num_chunks_per_group+i], (i+1)*chunks[j*num_chunks_per_group+i], j*num_chunks_per_group+i); //(int)(n-log2((double)num_chunks))
            // contract_tensor<<<num_chunks_per_group, (int)(pow(2, qubits_per_chunk)) , sharedMemSize, streams[j*num_chunks_per_group+i]>>>(state_d[j*num_chunks_per_group+i], H_d[0], 1, new_idx_d[j*num_chunks_per_group+i], old_idx_d[j*num_chunks_per_group+i], qubits_per_chunk, i*chunks[j*num_chunks_per_group+i], (i+1)*chunks[j*num_chunks_per_group+i], j*num_chunks_per_group+i); //(int)(n-log2((double)num_chunks))

            // ### Here we run Grover's algorithm
            applyGateAllQubits(
                state_d[j*num_chunks_per_group+i],
                H_d[0], new_idx_d[j*num_chunks_per_group+i],
                old_idx_d[j*num_chunks_per_group+i], qubits_per_chunk,
                dimBlock,
                dimGrid,
                sharedMemSize, i*chunks[j*num_chunks_per_group+i],
                (i+1)*chunks[j*num_chunks_per_group+i],
                streams[j*num_chunks_per_group+i]
            );
            for (int l = 0; l < k; ++l) {
                if (oracle_chunk == (j*num_chunks_per_group+i)) {
                    // printf("applyPhaseFlip at: %d\n", (j*num_chunks_per_group+i));
                    applyPhaseFlip<<<dimGrid, dimBlock, 0, streams[j*num_chunks_per_group+i]>>>(state_d[j*num_chunks_per_group+i], markedState);
                }

                applyDiffusionOperator(
                    state_d[j*num_chunks_per_group+i],
                    X_H_d[0], H_d[0], X_d[0], Z_d[0], new_idx_d[j*num_chunks_per_group+i],
                    old_idx_d[j*num_chunks_per_group+i], qubits_per_chunk, dimBlock, dimGrid, sharedMemSize,
                    i*chunks[j*num_chunks_per_group+i], (i+1)*chunks[j*num_chunks_per_group+i],
                    streams[j*num_chunks_per_group+i]
                );
            }
            hipMemcpyAsync(state_h[j*num_chunks_per_group+i], state_d[j*num_chunks_per_group+i], chunks[j*num_chunks_per_group+i] * sizeof(Complex), hipMemcpyDeviceToHost, streams[j*num_chunks_per_group+i]);
            hipStreamSynchronize(streams[i]);
            hipFree(state_d[j*num_chunks_per_group+i]);
            // hipHostFree(state_h[j*num_chunks_per_group+i]);
            // hipStreamDestroy(streams[i]);
        }

        // for (int i = 0; i < num_chunks_per_group; ++i){
        //     hipStreamSynchronize(streams[j*num_chunks_per_group+i]);
        //     // hipFree(state_d[j*num_chunks_per_group+i]);

        // }
    }

    for (int i = 0; i < num_chunks; ++i) {

        // hipStreamSynchronize(streams[i]);
        // hipFree(state_d[j*num_chunks_per_group+i]);
        hipStreamDestroy(streams[i]);
    }

    double elapsed = omp_get_wtime() - time;
    printf("Time: %f \n", elapsed);


    // hipMemcpy(state_h, state_d, N * sizeof(Complex), hipMemcpyDeviceToHost);

    for (int i = 0; i < num_chunks; ++i) {
        printf("chunk id: %d ######################################\n", i);
        printState(state_h[i], chunks[i], "Initial state");
    }





    for (int i = 0; i < num_devices; ++i) {
        hipFree(H_d[i]);
        hipFree(I_d[i]);
        hipFree(Z_d[i]);
        hipFree(X_d[i]);
        hipFree(X_H_d[i]);
    }


    for (int i = 0; i < num_chunks; ++i) {
        hipFree(state_d[i]);
        hipHostFree(state_h[i]);
        hipFree(new_idx_d[i]);
        hipFree(old_idx_d[i]);
    }

    return 0;
}
