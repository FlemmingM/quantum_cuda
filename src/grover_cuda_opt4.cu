#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <math.h>
#include <string.h>
#include <errno.h>
#include <omp.h>
#include "utils_cuda_opt4.h"

typedef hipDoubleComplex Complex;

int main(int argc, char* argv[]) {

    // collect input args
    // if (argc < 6) {
    //     fprintf(stderr, "Usage: %s n qubits<int>; marked state<int>; number of samples<int>; fileName<string>; verbose 0 or 1<int>\n", argv[0]);
    //     return 1;
    // }

    int n = atoi(argv[1]);
    long long int N = (long long int)pow(2, n);
    long long int markedState = atoi(argv[2]);
    const int chunk_size = atoi(argv[3]);
    // const char* fileName = argv[4];
    // int verbose = atoi(argv[5]);

    if (markedState > (N-1)) {
        fprintf(stderr, "You chose a markedState %d but the largest state possible is state %d", markedState, (N-1));
        return 1;
    }


    // Define the number of groups to do the parallel search with more than 10 qubits
    // while still using the fast shared memory

    int num_groups = pow(2, ((n - 10 < 0) ? 0 : (n - 10)));
    printf("num_groups: %d\n", num_groups);
    // printf("test: %d\n", log2)

    int num_chunks = num_groups * chunk_size;
    int qubits_per_chunk = (int)(n-log2((double)num_chunks));
    printf("num chunks: %d, n per chunk: %d\n",num_chunks, qubits_per_chunk);

    // Define the config for threads, devices and streams
    // const int num_chunks = 2;



    int chunks[num_chunks];
    // int sharedMemSizes[num_chunks];
    int sharedMemSize = (int)(pow(2, 10)) * sizeof(Complex);
    // if (n > 10) {
    //     int sharedMemSize = (int)(pow(2, 10)) * sizeof(Complex);
    // } else {
    //     int sharedMemSize = N * sizeof(Complex);
    // }


    for (int i = 0; i < num_chunks; ++i) {
        chunks[i] = N / num_chunks;
        // sharedMemSizes[i] = N / num_chunks * sizeof(Complex);
    }

    int val = 1024;
    dim3 dimBlock(val);
    dim3 dimGrid((N/num_chunks + dimBlock.x - 1) / dimBlock.x);

    // Set the gates:
    int num_devices = 1;
    Complex *H_d[num_devices];
    Complex *I_d[num_devices];
    Complex *Z_d[num_devices];
    Complex *X_d[num_devices];
    Complex *X_H_d[num_devices];
    allocateGatesDevice(num_devices, H_d, I_d, Z_d, X_d, X_H_d);

    // Init the arrays
    Complex *state_h[num_chunks];
    Complex *state_d[num_chunks];
    int *new_idx_d[num_chunks];
    int *old_idx_d[num_chunks];

    for (int i = 0; i < num_chunks; ++i) {
         // Init the state
        hipHostMalloc((void **)&state_h[i], chunks[i] * sizeof(Complex));
        hipMalloc((void **)&state_d[i], chunks[i] * sizeof(Complex));
        // Init the |0>^(xn) state and the new_state
        state_h[i][0] = make_hipDoubleComplex(1.0, 0.0);
        for (int j = 1; j < chunks[i]; ++j) {
            state_h[i][j] = make_hipDoubleComplex(0.0, 0.0);
        }
        // hipMemcpy(state_d[i], state_h[i], chunks[i] * sizeof(Complex), hipMemcpyHostToDevice);

        // Malloc the indices on the device
        hipMalloc(&new_idx_d[i], dimGrid.x * dimBlock.x * qubits_per_chunk * sizeof(int));
        hipMalloc(&old_idx_d[i], dimGrid.x * dimBlock.x * qubits_per_chunk * sizeof(int));
    }



    hipStream_t streams[chunk_size];


    // Assuming we have t = 1 solution in grover's algorithm
    // we have k = floor(pi/4 * sqrt(N/num_chunks))
    int k = (int)floor(M_PI / 4 * sqrt(N/num_chunks));



    double time = omp_get_wtime();


    int count = 0;
    for (int j = 0; j < num_groups; ++j) {
        #pragma omp parallel for num_threads(chunk_size)
        for (int i = 0; i < chunk_size; ++i) {
            hipStreamCreate(&streams[i]);
            hipMemcpyAsync(state_d[j*chunk_size+i], state_h[j*chunk_size+i], chunks[j*chunk_size+i] * sizeof(Complex), hipMemcpyHostToDevice, streams[i]);
            contract_tensor<<<dimGrid, dimBlock, sharedMemSize>>>(state_d[j*chunk_size+i], H_d[0], 0, new_idx_d[j*chunk_size+i], old_idx_d[j*chunk_size+i], qubits_per_chunk, i*chunks[j*chunk_size+i], (i+1)*chunks[j*chunk_size+i]); //(int)(n-log2((double)num_chunks))
            hipMemcpyAsync(state_h[j*chunk_size+i], state_d[j*chunk_size+i], chunks[j*chunk_size+i] * sizeof(Complex), hipMemcpyDeviceToHost, streams[i]);
            hipStreamSynchronize(streams[i]);
            hipStreamDestroy(streams[i]);

            count += 1;
        }
    }



    // contract_tensor<<<gridSize, blockSize, sharedMemSize>>>(state_d, H_d, 0, new_idx_d, old_idx_d, n, N);
    // contract_tensor<<<gridSize, blockSize>>>(state_d, H_d, 0, new_state_d, shape_d, new_idx_d, old_idx_d, n, N);

        // contract_tensor_baseline<<<dimGrid, dimBlock>>>(state, gate, i, new_state, shape, n, N);
        // hipDeviceSynchronize();
        // Update the state with the new state
    // updateState<<<gridSize, blockSize>>>(state_d, new_state_d, N);





    // Now apply the H gate n times, once for each qubit
    // applyGateAllQubits(state_d, H_d, new_idx_d, old_idx_d, n, N, dimBlock, dimGrid, sharedMemSize);


    // for (int i = 0; i < k; ++i) {
    //     applyPhaseFlip<<<dimGrid, dimBlock>>>(state_d, markedState);
    //     applyDiffusionOperator(state_d, H_d, X_d, Z_d, new_idx_d, old_idx_d, n, N, dimBlock, dimGrid, sharedMemSize);
    //     // hipDeviceSynchronize();
    // }

    // hipDeviceSynchronize();
    double elapsed = omp_get_wtime() - time;
    printf("Time: %f \n", elapsed);


    // hipMemcpy(state_h, state_d, N * sizeof(Complex), hipMemcpyDeviceToHost);

    for (int i = 0; i < num_chunks; ++i) {
        printf("chunk id: %d ######################################\n", i);
        printState(state_h[i], chunks[i], "Initial state");
    }





    for (int i = 0; i < num_devices; ++i) {
        hipFree(H_d[i]);
        hipFree(I_d[i]);
        hipFree(Z_d[i]);
        hipFree(X_d[i]);
        hipFree(X_H_d[i]);
    }


    // free(H_h);
    // free(I_h);
    // free(Z_h);
    // free(X_h);
    // free(X_H_h);

    for (int i = 0; i < num_chunks; ++i) {
        hipFree(state_d[i]);
        hipHostFree(state_h[i]);
        hipFree(new_idx_d[i]);
        hipFree(old_idx_d[i]);
    }

    return 0;
}
