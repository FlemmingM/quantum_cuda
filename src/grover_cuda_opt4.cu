#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <math.h>
#include <string.h>
#include <errno.h>
#include <omp.h>
#include "utils_cuda_opt4.h"

typedef hipDoubleComplex Complex;


#define cudaCheckError(call) {                               \
    hipError_t err = call;                                  \
    if (err != hipSuccess) {                                \
        fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n", \
                __FILE__, __LINE__, hipGetErrorString(err));\
        exit(EXIT_FAILURE);                                  \
    }                                                        \
}



int main(int argc, char* argv[]) {

    // collect input args
    // if (argc < 6) {
    //     fprintf(stderr, "Usage: %s n qubits<int>; marked state<int>; number of samples<int>; fileName<string>; verbose 0 or 1<int>\n", argv[0]);
    //     return 1;
    // }

    int n = atoi(argv[1]);
    long long int N = (long long int)pow(2, n);
    long long int markedState = atoi(argv[2]);
    const int num_chunks_per_group = N / 1024;
    // const int num_chunks_per_group = atoi(argv[3]);
    // const int block_size = atoi(argv[4]);
    // const char* fileName = argv[4];
    // int verbose = atoi(argv[5]);

    if (markedState > (N-1)) {
        fprintf(stderr, "You chose a markedState %d but the largest state possible is state %d", markedState, (N-1));
        return 1;
    }


    // Define the number of groups to do the parallel search with more than 10 qubits
    // while still using the fast shared memory

    long long int num_groups = (long long int)pow(2, ((n - 12 < 0) ? 0 : (n - 12)));
    printf("num_groups: %d\n", num_groups);
    printf("num_chunks_per_group: %d\n", num_chunks_per_group);
    // printf("test: %d\n", log2)
    long long int num_chunks = num_groups * num_chunks_per_group;
    // int qubits_per_chunk = (int)(n-log2((double)num_chunks));
    int qubits_per_chunk = 10;
    // long long int N_group = (int)(pow(2, qubits_per_chunk));
    int N_group = 1024;

    printf("num chunks: %lld, n per chunk: %lld\n",num_chunks, qubits_per_chunk);

    // Define the config for threads, devices and streams
    // const int num_chunks = 2;

    // int chunks[num_chunks];
    // int sharedMemSizes[num_chunks];
    int sharedMemSize = (int)(pow(2, 11)) * sizeof(Complex);
    printf("Using shared memory size: %d, per group[%d]: %d\n", sharedMemSize, num_chunks_per_group, sharedMemSize/num_chunks_per_group);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    if (sharedMemSize > deviceProp.sharedMemPerBlock) {
        printf("Requested shared memory size (%d bytes) exceeds the device limit (%d bytes).\n", sharedMemSize, deviceProp.sharedMemPerBlock);
        return -1;
    }

    // for (int i = 0; i < num_chunks; ++i) {
    //     chunks[i] = N / num_chunks;
    // }


    printf("Using chunk size for computation: %d\n", N_group);

    // Define the chunk for the oracle
    printf("N: %lld, markedState: %d, num_chunks: %lld\n", N, markedState, num_chunks);
    printf("%f\n", (N / num_chunks));

    // long long int oracle_chunk = markedState / (N / num_chunks);
    // long long int oracle_chunk = markedState / (N / num_chunks);

    printf("helooooooooo\n");
    // markedState = markedState % (N / num_chunks);
    // long long int recoveredState = oracle_chunk*(N / num_chunks)+markedState;
    // printf("oracle_chunk: %lld, pos: %lld, recovered: %lld\n", oracle_chunk, markedState, recoveredState);

    // dim3 dimBlock(block_size);
    // dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x);

    // dim3 dimBlock(N_group);

    dim3 dimBlock(1024);
    dim3 dimGrid(num_chunks_per_group);

    printf("dimGrid: %lld, dimBlock: %lld\n", num_chunks_per_group, N_group);

    // Set the gates:
    int num_devices = 1;
    Complex *H_d[num_devices];
    Complex *I_d[num_devices];
    Complex *Z_d[num_devices];
    Complex *X_d[num_devices];
    Complex *X_H_d[num_devices];
    allocateGatesDevice(num_devices, H_d, I_d, Z_d, X_d, X_H_d);






    // TODO make new_idx and old_idx only for num_chunks_per_group
    // for (int i = 0; i < num_chunks; ++i) {
    //      // Init the state
    //     // hipHostMalloc((void **)&state_h[i], chunks[i] * sizeof(Complex));
    //     // // hipMalloc((void **)&state_d[i], chunks[i] * sizeof(Complex));
    //     // // Init the |0>^(xn) state and the new_state
    //     // state_h[i][0] = make_hipDoubleComplex(1.0, 0.0);
    //     // for (int j = 1; j < chunks[i]; ++j) {
    //     //     state_h[i][j] = make_hipDoubleComplex(0.0, 0.0);
    //     // }
    //     // hipMemcpy(state_d[i], state_h[i], chunks[i] * sizeof(Complex), hipMemcpyHostToDevice);

    //     // Malloc the indices on the device
    //     // hipMalloc(&new_idx_d[i], dimGrid.x * dimBlock.x * qubits_per_chunk * sizeof(int));
    //     // hipMalloc(&old_idx_d[i], dimGrid.x * dimBlock.x * qubits_per_chunk * sizeof(int));
    //     hipMalloc(&new_idx_d[i], (int)(pow(2, qubits_per_chunk)) * qubits_per_chunk * sizeof(int));
    //     hipMalloc(&old_idx_d[i], (int)(pow(2, qubits_per_chunk)) * qubits_per_chunk * sizeof(int));
    // }




    // Init the arrays
    // Complex *state_h[num_chunks];
    // Complex *state_d[num_chunks];
    // printf("Hello there!!!!\n");
    // int *new_idx_d[num_chunks_per_group];
    // int *old_idx_d[num_chunks_per_group];



    // // Assuming we have t = 1 solution in grover's algorithm
    // // we have k = floor(pi/4 * sqrt(N/num_chunks))
    long long int k = (int)floor(M_PI / 4 * sqrt(N/num_chunks));
    printf("running %lld rounds\n", k);


    double time = omp_get_wtime();

    hipStream_t streams[num_chunks_per_group];

    Complex *state_h[num_chunks_per_group];
    Complex *state_d[num_chunks_per_group];
    printf("Hello there!!!!\n");
    int *new_idx_d[num_chunks_per_group];
    int *old_idx_d[num_chunks_per_group];
    // for (int i = 0; i < num_chunks; ++i) {
    //     cudaCheckError(hipMalloc(&new_idx_d[i], N_group * qubits_per_chunk * sizeof(int)));
    //     cudaCheckError(hipMalloc(&old_idx_d[i], N_group * qubits_per_chunk * sizeof(int)));
    //     // cudaCheckError(hipStreamCreate(&streams[i]));
    //     // hipMalloc((void **)&state_d[index], chunks[index] * sizeof(Complex));
    // }

    // #pragma omp parallel for collapse(2) num_threads(num_groups * num_chunks_per_group)
    // for (int j = 0; j < num_groups; ++j) {
    //     for (int i = 0; i < num_chunks_per_group; ++i) {
    //         hipMalloc((void **)&state_d[index], chunks[index] * sizeof(Complex));
    //     }
    // }

    for (int j = 0; j < 1; ++j) {
        // printf("%d / %d\n", j, num_groups);
        // #pragma omp parallel for num_threads(num_groups)
        for (int i = 0; i < num_chunks_per_group; ++i) {
            // hipStreamCreate(&streams[i]);
            int index = j*num_chunks_per_group+i;
            cudaCheckError(hipStreamCreate(&streams[i]));
            // tid = omp_get_thread_num();
            // printf("Welcome to GFG from thread = %d\n", tid);


            // printf("j %d, i %d, num_chunks_per_group %d, index %d\n", j, i, num_chunks_per_group, index);

            cudaCheckError(hipHostMalloc((void **)&state_h[i], N_group * sizeof(Complex)));
            state_h[i][0] = make_hipDoubleComplex(1.0, 0.0);
            for (int idx = 1; idx < N_group; ++idx) {
                state_h[i][idx] = make_hipDoubleComplex(0.0, 0.0);
            }

            cudaCheckError(hipMalloc(&new_idx_d[i], N_group * qubits_per_chunk * sizeof(int)));
            cudaCheckError(hipMalloc(&old_idx_d[i], N_group * qubits_per_chunk * sizeof(int)));
            cudaCheckError(hipMalloc((void **)&state_d[i], N_group * sizeof(Complex)));
            cudaCheckError(hipMemcpyAsync(state_d[i], state_h[i], N_group * sizeof(Complex), hipMemcpyHostToDevice, streams[i]));
            // contract_tensor<<<num_chunks_per_group, N_group , sharedMemSize, streams[index]>>>(state_d[i], H_d[0], 0, new_idx_d[index], old_idx_d[index], qubits_per_chunk, i*N_group, (i+1)*N_group, index); //(int)(n-log2((double)num_chunks))
            // contract_tensor<<<num_chunks_per_group, N_group , sharedMemSize, streams[index]>>>(state_d[i], H_d[0], 1, new_idx_d[index], old_idx_d[index], qubits_per_chunk, i*N_group, (i+1)*N_group, index); //(int)(n-log2((double)num_chunks))

            // ### Here we run Grover's algorithm
            applyGateAllQubits(
                state_d[i],
                H_d[0], new_idx_d[i],
                old_idx_d[i], qubits_per_chunk,
                dimBlock,
                dimGrid,
                sharedMemSize,
                // i*N_group,
                // (i+1)*N_group,
                0,
                1024,
                streams[i]
            );
            // for (int l = 0; l < k; ++l) {
            //     if (oracle_chunk == (index)) {
            //         // printf("applyPhaseFlip at: %d\n", (index));
            //         applyPhaseFlip<<<dimGrid, dimBlock, 0, streams[i]>>>(state_d[i], markedState);
            //     }

            //     applyDiffusionOperator(
            //         state_d[i],
            //         X_H_d[0], H_d[0], X_d[0], Z_d[0], new_idx_d[i],
            //         old_idx_d[i], qubits_per_chunk, dimBlock, dimGrid, sharedMemSize,
            //         i*N_group, (i+1)*N_group,
            //         streams[i]
            //     );
            // }
            cudaCheckError(hipMemcpyAsync(state_h[i], state_d[i], N_group * sizeof(Complex), hipMemcpyDeviceToHost, streams[i]));
            cudaCheckError(hipStreamSynchronize(streams[i]));
            // cudaCheckError(hipFree(old_idx_d[i]));
            // cudaCheckError(hipFree(new_idx_d[i]));
            cudaCheckError(hipFree(state_d[i]));
            // cudaCheckError(hipHostFree(state_h[i]));
            cudaCheckError(hipStreamDestroy(streams[i]));
        }

        // for (int i = 0; i < num_chunks_per_group; ++i){
        //     printf("chunk id: %d ######################################\n", i);
        //     printState(state_h[i], N_group, "Initial state");
        //     // hipFree(state_d[j*2 + i]);
        //     // hipFree(old_idx_d[j*2 + i]);
        //     // hipFree(new_idx_d[j*2 + i]);
        //     // hipHostFree(state_h[j*2 + i]);
        // }
    }

    // for (int i = 0; i < num_chunks_per_group; ++i) {

    //     // hipStreamSynchronize(streams[i]);
    //     // hipFree(state_d[index]);
    //     cudaCheckError(hipStreamDestroy(streams[i]));
    // }

    double elapsed = omp_get_wtime() - time;
    printf("Time: %f \n", elapsed);


    // hipMemcpy(state_h, state_d, N * sizeof(Complex), hipMemcpyDeviceToHost);

    // for (int i = 0; i < num_chunks; ++i) {
        // printf("chunk id: %d ######################################\n", i);
    printState(state_h[num_chunks_per_group-1], N_group, "Initial state");
    // }





    for (int i = 0; i < num_devices; ++i) {
        hipFree(H_d[i]);
        hipFree(I_d[i]);
        hipFree(Z_d[i]);
        hipFree(X_d[i]);
        hipFree(X_H_d[i]);
    }


    // for (int i = 0; i < num_chunks; ++i) {
    //     // hipFree(state_d[i]);
    //     // hipHostFree(state_h[i]);
    //     hipFree(new_idx_d[i]);
    //     hipFree(old_idx_d[i]);
    // }

    return 0;
}
