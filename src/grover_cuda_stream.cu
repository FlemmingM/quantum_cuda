#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <math.h>
#include <string.h>
#include <errno.h>
#include <omp.h>
#include "utils_cuda.h"
#include "utils_cuda_stream.h"


typedef hipDoubleComplex Complex;



int main(int argc, char* argv[]) {

    // collect input args
    // if (argc < 6) {
    //     fprintf(stderr, "Usage: %s n qubits<int>; marked state<int>; number of samples<int>; fileName<string>; verbose 0 or 1<int>\n", argv[0]);
    //     return 1;
    // }

    int n = atoi(argv[1]);
    long long int N = (long long int)pow(2, n);
    long long int markedState = atoi(argv[2]);
    const int num_chunks_per_group = atoi(argv[3]);
    const int num_qubits_per_group = atoi(argv[4]);
    // const int block_size = atoi(argv[4]);
    // const char* fileName = argv[4];
    // int verbose = atoi(argv[5]);

    if (markedState > (N-1)) {
        fprintf(stderr, "You chose a markedState %d but the largest state possible is state %d", markedState, (N-1));
        return 1;
    }


    // Define the number of groups to do the parallel search with more than 10 qubits
    // while still using the fast shared memory

    long long int num_groups = N / pow(2, num_qubits_per_group);
    int num_qubits_per_chunk = num_qubits_per_group - (int)log2(num_chunks_per_group);
    int N_chunk = pow(2, num_qubits_per_chunk);
    long long int num_chunks = num_groups * num_chunks_per_group;

    if (N_chunk > pow(2, 10)) {
        fprintf(stderr, "You chose a number of qubits per group of: %d and a number of chunks per group of: %d\n Change the config so that the number of qubits per chunk is maximally 10 to fit into 1 block", num_qubits_per_group, num_chunks_per_group);
        return 1;
    }

    int sharedMemSize = (int)(pow(2, 11)) * sizeof(Complex);


    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    if (sharedMemSize > deviceProp.sharedMemPerBlock) {
        printf("Requested shared memory size (%d bytes) exceeds the device limit (%d bytes).\n", sharedMemSize, deviceProp.sharedMemPerBlock);
        return -1;
    }


    long long int oracle_chunk = markedState / (N / num_chunks);


    markedState = markedState % (N / num_chunks);
    long long int recoveredState = oracle_chunk*(N / num_chunks)+markedState;


    dim3 dimBlock(N_chunk);
    dim3 dimGrid(1);
    // dim3 dimGrid(num_chunks_per_group);

    int print_val = 0;
    if (print_val == 1) {
        printf("N: %lld\n", N);
        printf("n: %d\n", n);
        printf("num_groups: %lld\n", num_groups);
        printf("num_chunks_per_group: %d\n", num_chunks_per_group);
        printf("num_qubits_per_chunk: %d\n", num_qubits_per_chunk);
        printf("N_chunk: %d\n", N_chunk);
        printf("num_chunks: %lld\n", num_chunks);
        printf("oracle_chunk: %lld, pos: %lld, recovered: %lld\n", oracle_chunk, markedState, recoveredState);
        printf("dimGrid: %d, dimBlock: %d\n", dimGrid.x, dimBlock.x);
    }


    // Set the gates:
    int num_devices = 1;
    Complex *H_d[num_devices];
    Complex *I_d[num_devices];
    Complex *Z_d[num_devices];
    Complex *X_d[num_devices];
    Complex *X_H_d[num_devices];
    allocateGatesDevice(num_devices, H_d, I_d, Z_d, X_d, X_H_d);



    // // Assuming we have t = 1 solution in grover's algorithm
    // // we have k = floor(pi/4 * sqrt(N/num_chunks))
    long long int k = (int)floor(M_PI / 4 * sqrt(N/num_chunks));
    // printf("running %lld rounds\n", k);



    double time = omp_get_wtime();

    hipStream_t streams[num_chunks_per_group];

    Complex *solution_state_h;
    Complex *state_h[num_chunks_per_group];
    Complex *state_d[num_chunks_per_group];
    int *new_idx_d[num_chunks_per_group];
    int *old_idx_d[num_chunks_per_group];

    // To get the parallel search results
    int *d_maxIndex;
    int *h_maxIndex;
    int *d_chunk_ids;
    int *h_chunk_ids;
    double *d_maxValue;
    double *h_maxValue;
    hipMalloc(&d_maxIndex, num_chunks_per_group*sizeof(int));
    hipHostMalloc(&h_maxIndex, num_chunks_per_group*sizeof(int));
    hipMalloc(&d_maxValue, num_chunks_per_group*sizeof(double));
    hipHostMalloc(&h_maxValue, num_chunks_per_group*sizeof(double));
    hipMalloc(&d_chunk_ids, num_chunks_per_group*sizeof(double));
    hipHostMalloc(&h_chunk_ids, num_chunks_per_group*sizeof(double));


    // init the arrays:



    // Create the streams
    for (int i = 0; i < num_chunks_per_group; ++i) {
        hipStreamCreate(&streams[i]);
        hipHostMalloc((void **)&state_h[i], N_chunk * sizeof(Complex));
        state_h[i][0] = make_hipDoubleComplex(1.0, 0.0);
        for (int idx = 1; idx < N_chunk; ++idx) {
            state_h[i][idx] = make_hipDoubleComplex(0.0, 0.0);
        }
        hipMalloc(&new_idx_d[i], N_chunk * num_qubits_per_chunk * sizeof(int));
        hipMalloc(&old_idx_d[i], N_chunk * num_qubits_per_chunk * sizeof(int));
        hipMalloc((void **)&state_d[i], N_chunk * sizeof(Complex));
        hipMemcpyAsync(state_d[i], state_h[i], N_chunk * sizeof(Complex), hipMemcpyHostToDevice, streams[i]);
    }


    // allocate the solution state:
    hipHostMalloc((void **)&solution_state_h, N_chunk * sizeof(Complex));

    int marked_chunk = -99;
    int marked_max_val = -99;
    int marked_max_idx = -99;
    for (int j = 0; j < num_groups; ++j) {
        // printf("%d / %d\n", j, num_groups);
        // #pragma omp parallel for num_threads(num_chunks_per_group)
        for (int i = 0; i < num_chunks_per_group; ++i) {
            // hipStreamCreate(&streams[i]);
            int index = j*num_chunks_per_group+i;

            // ### Here we run Grover's algorithm
            // initState<<<dimGrid, dimBlock, 0, streams[i]>>>(state_d[i], N_chunk);
            applyGateAllQubits(
                state_d[i],
                H_d[0], new_idx_d[i],
                old_idx_d[i], num_qubits_per_chunk,
                dimBlock,
                dimGrid,
                sharedMemSize,
                0,
                N_chunk,
                streams[i]
            );
            for (int l = 0; l < k; ++l) {
                if (oracle_chunk == (index)) {
                    // printf("oracle chunk_id: %d\n", index);
                    applyPhaseFlip<<<dimGrid, dimBlock, 0, streams[i]>>>(state_d[i], markedState);
                }

                applyDiffusionOperator(
                    state_d[i],
                    X_H_d[0], H_d[0], X_d[0], Z_d[0], new_idx_d[i],
                    old_idx_d[i], num_qubits_per_chunk, dimBlock, dimGrid, sharedMemSize,
                    0, N_chunk,
                    streams[i]
                );
            }
        }

        for (int i = 0; i < num_chunks_per_group; ++i) {
            hipStreamSynchronize(streams[i]);
        }
        hipDeviceSynchronize();


        for (int i = 0; i < num_chunks_per_group; ++i){
            findMaxIndexKernel<<<1, N_chunk, 0, streams[i]>>>(state_d[i], d_maxIndex, d_maxValue, N_chunk, i, d_chunk_ids);
            hipMemcpyAsync(h_maxIndex, d_maxIndex, num_chunks_per_group*sizeof(int), hipMemcpyDeviceToHost, streams[i]);
            hipMemcpyAsync(h_chunk_ids, d_chunk_ids, num_chunks_per_group*sizeof(int), hipMemcpyDeviceToHost, streams[i]);
            hipMemcpyAsync(h_maxValue, d_maxValue, num_chunks_per_group*sizeof(double), hipMemcpyDeviceToHost, streams[i]);

        }

        for (int i = 0; i < num_chunks_per_group; ++i) {
            hipStreamSynchronize(streams[i]);
        }
        hipDeviceSynchronize();


        for (int i = 0; i < num_chunks_per_group; ++i){
            // printf("chunk id: %d, maxIdx: %d, maxVal: %f\n", h_chunk_ids[i], h_maxIndex[i], h_maxValue[i]);
            if(h_maxValue[i] >= 0.5){
                // printf("Solution: chunk id: %d, maxIdx: %d, maxVal: %f\n", h_chunk_ids[i], h_maxIndex[i], h_maxValue[i]);
                marked_chunk = h_chunk_ids[i];
                marked_max_idx = h_maxIndex[i];
                marked_max_val = h_maxValue[i];
                int index = marked_chunk % num_chunks_per_group;

                hipMemcpyAsync(solution_state_h, state_d[index], N_chunk * sizeof(Complex), hipMemcpyDeviceToHost, streams[index]);
                hipStreamSynchronize(streams[index]);
            }
        }
        hipDeviceSynchronize();

        for (int i = 0; i < num_chunks_per_group; ++i){
            initState<<<dimGrid, dimBlock, 0, streams[i]>>>(state_d[i], N_chunk);
            hipStreamSynchronize(streams[i]);
        }
        hipDeviceSynchronize();

    } // end of the out loop

    for (int i = 0; i < num_chunks_per_group; ++i) {
        hipStreamDestroy(streams[i]);
    }

    double elapsed = omp_get_wtime() - time;
    // printf("Time: %f \n", elapsed);
    // n, k, num_groups, num_chunks, n_per_group, chunks_per_group, num_threads, marked_chunk, markedState, marked_max_idx, marked_max_val, time
    printf("%d,%lld,%lld,%lld,%d,%d,%d,%d,%d,%d,%f,%f\n",
        n, k, num_groups, num_chunks, num_qubits_per_group, num_chunks_per_group, dimBlock.x, marked_chunk, markedState, marked_max_idx, marked_max_val, elapsed);

    // printState(solution_state_h, N_chunk, "Initial state");


    for (int i = 0; i < num_devices; ++i) {
        hipFree(H_d[i]);
        hipFree(I_d[i]);
        hipFree(Z_d[i]);
        hipFree(X_d[i]);
        hipFree(X_H_d[i]);
    }


    for (int i = 0; i < num_chunks_per_group; ++i) {
        hipFree(new_idx_d[i]);
        hipFree(old_idx_d[i]);
        hipFree(state_d[i]);
        hipHostFree(state_h[i]);


    }

    hipHostFree(solution_state_h);
    hipFree(d_maxIndex);
    hipFree(d_chunk_ids);
    hipFree(d_maxValue);
    hipHostFree(h_maxIndex);
    hipHostFree(h_chunk_ids);
    hipHostFree(h_maxValue);

    return 0;
}
