
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <math.h>
#include <string.h>
#include <errno.h>
#include <omp.h>

typedef hipDoubleComplex Complex;


void allocateGatesDevice(const int num_devices, Complex **H_d, Complex **I_d, Complex **Z_d, Complex **X_d, Complex **X_H_d) {

    // Define the gates
    hipDoubleComplex H_h[4] = {
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0),
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(-1.0 / sqrt(2.0), 0.0)
    };
    hipDoubleComplex X_H_h[4] = {
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(-1.0 / sqrt(2.0), 0.0),
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0)
    };
    hipDoubleComplex I_h[4] = {
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0),
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(1.0, 0.0)
    };
    hipDoubleComplex Z_h[4] = {
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0),
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(-1.0, 0.0)
    };
    hipDoubleComplex X_h[4] = {
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(1.0, 0.0),
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0)
    };

    for (int i = 0; i < num_devices; i++) {
        // Set the device
        hipSetDevice(i);

        // Malloc the gate on device
        hipMalloc((void **)&H_d[i], 4 * sizeof(Complex));
        hipMalloc((void **)&X_H_d[i], 4 * sizeof(Complex));
        hipMalloc((void **)&I_d[i], 4 * sizeof(Complex));
        hipMalloc((void **)&Z_d[i], 4 * sizeof(Complex));
        hipMalloc((void **)&X_d[i], 4 * sizeof(Complex));

        // Copy from host to device
        hipMemcpy(H_d[i], H_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
        hipMemcpy(X_H_d[i], X_H_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
        hipMemcpy(I_d[i], I_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
        hipMemcpy(Z_d[i], Z_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
        hipMemcpy(X_d[i], X_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
    }
}


__global__ void findMaxIndexKernel(Complex* d_array, int* d_maxIndex, double* d_maxValue, int size, int chunk_id, int* chunk_ids) {
    __shared__ Complex sharedArray[1024];
    __shared__ int sharedIndex[1024];

    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size) {
        sharedArray[tid] = d_array[index];
        sharedIndex[tid] = index;
    } else {
        sharedArray[tid] = make_hipDoubleComplex(-99.0, 0.00);  // Set to minimum value if out of bounds
        sharedIndex[tid] = -1;        // Invalid index
    }

    __syncthreads();

    // Perform reduction to find the max value and its index
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride && index + stride < size) {
            if (hipCreal(sharedArray[tid]) < hipCreal(sharedArray[tid + stride])) {
                sharedArray[tid] = sharedArray[tid + stride];
                sharedIndex[tid] = sharedIndex[tid + stride];
            }
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tid == 0) {


        // printf("Val: %f, Index: %d, chunk_id: %d\n", cuCreal(sharedArray[0]), sharedIndex[0], chunk_id);
        // printf("Index: %d\n", sharedIndex[0]);
        // printf("chunk_id: %d\n", chunk_id);
        d_maxIndex[chunk_id] = sharedIndex[0];
        d_maxValue[chunk_id] = hipCreal(sharedArray[0]);
        chunk_ids[chunk_id] = chunk_id;
        // printf("Val: %f, Index: %d, chunk_id: %d\n", cuCreal(sharedArray[0]), sharedIndex[0], chunk_id);

        // for (int i = 0; i < 2; ++i){
        //     printf("d_maxIndex: %d\n", d_maxIndex[i]);
        // }

    }
}


__global__ void compute_idx(
        int qubit,
        int* new_idx,
        int* old_idx,
        const int n,
        const long long int N,
        int* old_linear_idxs
) {
    extern __shared__ int shared_memory[]; // Use shared memory

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = idx * n;
    int offset2 = qubit*2*N;

    if (idx < N) {
        int temp = idx;

        // Compute the multi-dimensional index
        for (int i = n - 1; i >= 0; --i) {
            new_idx[offset + i] = temp % 2;
            temp /= 2;
        }

        // Copy new_idx to old_idx
        for (int i = 0; i < n; ++i) {
            old_idx[offset + i] = new_idx[offset + i];
        }

        // Compute the two values for j = 0 and j = 1 and store in shared memory
        for (int j = 0; j < 2; ++j) {
            old_idx[offset + qubit] = j;

            // Compute the linear index for old_idx
            int old_linear_idx = 0;
            int factor = 1;
            for (int i = n - 1; i >= 0; --i) {
                old_linear_idx += old_idx[offset + i] * factor;
                factor *= 2;
            }
            // coalesced
            // old_linear_idx += base_idx;
            shared_memory[2*idx + j] = old_linear_idx;
        }
        // coalesced
        old_linear_idxs[2*idx + offset2] = shared_memory[2*idx];
        old_linear_idxs[2*idx + 1 + offset2] = shared_memory[2*idx+1];
    }
}

__global__ void contract_tensor(
        Complex* state,
        const Complex* gate,
        int qubit,
        int* new_idx,
        const int n,
        const long long int N,
        int* old_linear_idxs,
        const int chunk_size
) {
    extern __shared__ Complex shared_mem[]; // Use shared memory
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < N) {

        int offset = threadIdx.x * n;
        int base_idx = (idx / chunk_size) * chunk_size;

        Complex* buffer1 = shared_mem;
        Complex* buffer2 = shared_mem + chunk_size;

        // Compute the two values for j = 0 and j = 1 and store in shared memory
        for (int j = 0; j < 2; ++j) {
            // needed to translate back to the full state array!!!
            int old_linear_idx = old_linear_idxs[2*threadIdx.x + j + qubit*2*chunk_size];
            old_linear_idx += base_idx;


            if(j==0){
                buffer1[threadIdx.x] = hipCmul(gate[new_idx[offset + qubit] * 2 + j], state[old_linear_idx]);
            } else {
                buffer2[threadIdx.x] = hipCmul(gate[new_idx[offset + qubit] * 2 + j], state[old_linear_idx]);
            }

        }
        state[idx] = hipCadd(buffer1[threadIdx.x], buffer2[threadIdx.x]);
    }
}


__global__ void applyPhaseFlip(Complex* state, long long int idx) {
    state[idx] = hipCmul(state[idx], make_hipDoubleComplex(-1.0, 0.0));
}

__global__ void applyPhaseFlipParallel(Complex* state, const long long int N, const int N_chunk) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) {
        if ((idx % N_chunk) == N_chunk - 1) {
            state[idx] = hipCmul(state[idx], make_hipDoubleComplex(-1.0, 0.0));
        }
    }

}

void applyGateAllQubits(
    Complex* state,
    const Complex* gate,
    int* new_idx,
    int n,
    dim3 dimBlock,
    dim3 dimGrid,
    int sharedMemSize,
    const long long int N,
    int* old_linear_idxs,
    const int N_chunk,
    hipStream_t stream
    ) {

    for (int i = 0; i < n; ++i) {
        contract_tensor<<<dimGrid, dimBlock, sharedMemSize, stream>>>(state, gate, i, new_idx, n, N, old_linear_idxs, N_chunk);
    }
}

void applyGateSingleQubit(
    Complex* state,
    const Complex* gate,
    int* new_idx,
    int n,
    long long int idx,
    dim3 dimBlock,
    dim3 dimGrid,
    int sharedMemSize,
    const long long int N,
    int* old_linear_idxs,
    const int N_chunk,
    hipStream_t stream
    ) {

    contract_tensor<<<dimGrid, dimBlock, sharedMemSize, stream>>>(state, gate, idx, new_idx, n, N, old_linear_idxs, N_chunk);
}

void applyDiffusionOperator(
    Complex* state,
    const Complex* X_H,
    const Complex* H,
    const Complex* X,
    const Complex* Z,
    int* new_idx,
    int n,
    dim3 dimBlock,
    dim3 dimGrid,
    int sharedMemSize,
    const int N_chunk,
    const long long int N,
    int* old_linear_idxs,
    hipStream_t stream
    ) {
    applyGateAllQubits(state, X_H, new_idx, n, dimBlock, dimGrid, sharedMemSize, N, old_linear_idxs, N_chunk, stream);
    applyPhaseFlipParallel<<<dimGrid, dimBlock, 0, stream>>>(state, N, N_chunk);
    applyGateSingleQubit(state, Z, new_idx, n, 0, dimBlock, dimGrid, sharedMemSize, N, old_linear_idxs, N_chunk, stream);
    applyGateAllQubits(state, X, new_idx, n, dimBlock, dimGrid, sharedMemSize, N, old_linear_idxs, N_chunk, stream);
    applyGateSingleQubit(state, Z, new_idx, n, 0, dimBlock, dimGrid, sharedMemSize, N, old_linear_idxs, N_chunk, stream);
    applyGateAllQubits(state, H, new_idx, n, dimBlock, dimGrid, sharedMemSize, N, old_linear_idxs, N_chunk, stream);
}
