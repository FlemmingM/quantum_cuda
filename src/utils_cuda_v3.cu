
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <math.h>
#include <string.h>
#include <errno.h>
#include <omp.h>

typedef hipDoubleComplex Complex;


void allocateGatesDevice(const int num_devices, Complex **H_d, Complex **I_d, Complex **Z_d, Complex **X_d, Complex **X_H_d) {

    // Define the gates
    hipDoubleComplex H_h[4] = {
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0),
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(-1.0 / sqrt(2.0), 0.0)
    };
    hipDoubleComplex X_H_h[4] = {
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(-1.0 / sqrt(2.0), 0.0),
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0)
    };
    hipDoubleComplex I_h[4] = {
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0),
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(1.0, 0.0)
    };
    hipDoubleComplex Z_h[4] = {
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0),
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(-1.0, 0.0)
    };
    hipDoubleComplex X_h[4] = {
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(1.0, 0.0),
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0)
    };

    for (int i = 0; i < num_devices; i++) {
        // Set the device
        hipSetDevice(i);

        // Malloc the gate on device
        hipMalloc((void **)&H_d[i], 4 * sizeof(Complex));
        hipMalloc((void **)&X_H_d[i], 4 * sizeof(Complex));
        hipMalloc((void **)&I_d[i], 4 * sizeof(Complex));
        hipMalloc((void **)&Z_d[i], 4 * sizeof(Complex));
        hipMalloc((void **)&X_d[i], 4 * sizeof(Complex));

        // Copy from host to device
        hipMemcpy(H_d[i], H_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
        hipMemcpy(X_H_d[i], X_H_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
        hipMemcpy(I_d[i], I_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
        hipMemcpy(Z_d[i], Z_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
        hipMemcpy(X_d[i], X_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
    }
}


__global__ void findMaxIndexKernel(Complex* d_array, int* d_maxIndex, double* d_maxValue, int size, int chunk_id, int* chunk_ids) {
    __shared__ Complex sharedArray[1024];
    __shared__ int sharedIndex[1024];

    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size) {
        sharedArray[tid] = d_array[index];
        sharedIndex[tid] = index;
    } else {
        sharedArray[tid] = make_hipDoubleComplex(-99.0, 0.00);  // Set to minimum value if out of bounds
        sharedIndex[tid] = -1;        // Invalid index
    }

    __syncthreads();

    // Perform reduction to find the max value and its index
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride && index + stride < size) {
            if (hipCreal(sharedArray[tid]) < hipCreal(sharedArray[tid + stride])) {
                sharedArray[tid] = sharedArray[tid + stride];
                sharedIndex[tid] = sharedIndex[tid + stride];
            }
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tid == 0) {


        // printf("Val: %f, Index: %d, chunk_id: %d\n", cuCreal(sharedArray[0]), sharedIndex[0], chunk_id);
        // printf("Index: %d\n", sharedIndex[0]);
        // printf("chunk_id: %d\n", chunk_id);
        d_maxIndex[chunk_id] = sharedIndex[0];
        d_maxValue[chunk_id] = hipCreal(sharedArray[0]);
        chunk_ids[chunk_id] = chunk_id;
        // printf("Val: %f, Index: %d, chunk_id: %d\n", cuCreal(sharedArray[0]), sharedIndex[0], chunk_id);

        // for (int i = 0; i < 2; ++i){
        //     printf("d_maxIndex: %d\n", d_maxIndex[i]);
        // }

    }
}

__global__ void compute_idx(
        int qubit,
        int* new_idx,
        int* old_idx,
        const int n,
        const long long int N,
        int* old_linear_idxs
    ) {
    extern __shared__ int shared_memory[]; // Use shared memory

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = idx * n;
    // int chunk_size = pow(2, n);
    int offset2 = qubit*2*N;


    if (idx < N) {
        int temp = idx;

        // Compute the multi-dimensional index
        for (int i = n - 1; i >= 0; --i) {
            new_idx[offset + i] = temp % 2;
            temp /= 2;
        }

        // Copy new_idx to old_idx
        for (int i = 0; i < n; ++i) {
            old_idx[offset + i] = new_idx[offset + i];
        }

        // Compute the two values for j = 0 and j = 1 and store in shared memory
        for (int j = 0; j < 2; ++j) {
            old_idx[offset + qubit] = j;

            // Compute the linear index for old_idx
            int old_linear_idx = 0;
            int factor = 1;
            for (int i = n - 1; i >= 0; --i) {
                old_linear_idx += old_idx[offset + i] * factor;
                factor *= 2;
            }
            // coalesced
            shared_memory[2*idx + j] = old_linear_idx;
        }
        // coalesced
        old_linear_idxs[2*idx + offset2] = shared_memory[2*idx];
        old_linear_idxs[2*idx + 1 + offset2] = shared_memory[2*idx+1];
    }
}

__global__ void contract_tensor(
        Complex* state,
        const Complex* gate,
        int qubit,
        int* new_idx,
        const int n,
        const long long int N,
        int* old_linear_idxs

) {
    extern __shared__ Complex shared_mem[]; // Use shared memory
    int idx = blockDim.x * blockIdx.x + threadIdx.x;


    int chunk_size = pow(2, n);

    if (idx < N) {
        int idx2 = idx /2;
        int offset = (idx2 % chunk_size) * n;

        // Compute the two values for j = 0 and j = 1 and store in shared memory
        int j = idx % 2;

        int old_linear_idx = old_linear_idxs[2*(idx2 % chunk_size) + j + qubit*2*chunk_size];
        old_linear_idx += (idx2 / chunk_size) * chunk_size;

        Complex val = hipCmul(gate[new_idx[offset + qubit] * 2 + j], state[old_linear_idx]);
        // shared_mem[idx % chunk_size] = val;

        // works!
        shared_mem[idx % (2*chunk_size)] = val;

        // very slow!!!
        // shared_mem[(idx % (2*chunk_size)) + j*(chunk_size-1)] = val;
        // printf("idx: %d, idx2: %d, shared_mem_pos: %d, j: %d, old_lin_idx %d, val: %f\n", idx, idx2, idx % (2*chunk_size), j, old_linear_idx, cuCreal(val));

        __syncthreads();

        // needs to be adjusted for the blocks since we might have more than 1!!!
        val = shared_mem[idx % (2*chunk_size)];
        val.x += __shfl_down_sync(0xffffffff, val.x, 1);
        val.y += __shfl_down_sync(0xffffffff, val.y, 1);


        if ((idx % 2) == 0) {
            state[idx2] = val;
        }


        // very slow!!!
        // val = shared_mem[(idx % (2*chunk_size)) + j*(chunk_size-1)];
        // val.x += __shfl_down_sync(0xffffffff, val.x, chunk_size);
        // val.y += __shfl_down_sync(0xffffffff, val.y, chunk_size);
        // if (((idx/chunk_size) % 2) == 0) {
        //     state[idx2] = val;
        // }
    }
}


__global__ void applyPhaseFlip(Complex* state, long long int idx) {
    state[idx] = hipCmul(state[idx], make_hipDoubleComplex(-1.0, 0.0));
}

void applyGateAllQubits(
    Complex* state,
    const Complex* gate,
    int* new_idx,
    int n,
    dim3 dimBlock,
    dim3 dimGrid,
    int sharedMemSize,
    const long long int N,
    int* old_linear_idxs
    ) {

    for (int i = 0; i < n; ++i) {
        contract_tensor<<<dimGrid, dimBlock, sharedMemSize>>>(state, gate, i, new_idx, n, N, old_linear_idxs);
    }
}

void applyGateSingleQubit(
    Complex* state,
    const Complex* gate,
    int* new_idx,
    int n,
    long long int idx,
    dim3 dimBlock,
    dim3 dimGrid,
    int sharedMemSize,
    const long long int N,
    int* old_linear_idxs
    ) {

    contract_tensor<<<dimGrid, dimBlock, sharedMemSize>>>(state, gate, idx, new_idx, n, N, old_linear_idxs);
}

void applyDiffusionOperator(
    Complex* state,
    const Complex* X_H,
    const Complex* H,
    const Complex* X,
    const Complex* Z,
    int* new_idx,
    int n,
    dim3 dimBlock,
    dim3 dimGrid,
    int sharedMemSize,
    const int num_chunks_per_group,
    const long long int N_chunk,
    const long long int N,
    int* old_linear_idxs
    ) {
    applyGateAllQubits(state, X_H, new_idx, n, dimBlock, dimGrid, sharedMemSize, N, old_linear_idxs);
    for (int i = 0; i < num_chunks_per_group; ++i) {
        applyPhaseFlip<<<dimGrid, dimBlock, 0>>>(state, (i+1)*N_chunk - 1);
    }

    applyGateSingleQubit(state, Z, new_idx, n, 0, dimBlock, dimGrid, sharedMemSize, N, old_linear_idxs);
    applyGateAllQubits(state, X, new_idx, n, dimBlock, dimGrid, sharedMemSize, N, old_linear_idxs);
    applyGateSingleQubit(state, Z, new_idx, n, 0, dimBlock, dimGrid, sharedMemSize, N, old_linear_idxs);
    applyGateAllQubits(state, H, new_idx, n, dimBlock, dimGrid, sharedMemSize, N, old_linear_idxs);
}
