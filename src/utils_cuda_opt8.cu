
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <math.h>
#include <string.h>
#include <errno.h>
#include <omp.h>

typedef hipDoubleComplex Complex;

void printState(const Complex* state, long long int N, const char* message) {
    printf("%s\n", message);
    for (int i = 0; i < N; ++i) {
        printf("(%.15f + %.15fi) ", hipCreal(state[i]), hipCimag(state[i]));
    }
    printf("\n");
}

void saveArrayToCSV(const double *array, long long int N, const char* filename) {
    FILE *file = fopen(filename, "w");

    if (!file) {
        perror("Unable to open file");
        return;
    }
    fprintf(file, "position,probability\n");
    for (int i = 0; i < N; ++i) {
        fprintf(file, "pos%d,%f\n", i, array[i]);
    }
    fclose(file);
}


__device__ void AddComplex(hipDoubleComplex* a, hipDoubleComplex b){
  //transform the addresses of real and imag. parts to double pointers
  double *x = (double*)a;
  double *y = x+1;
  //use atomicAdd for double variables
  atomicAdd(x, hipCreal(b));
  atomicAdd(y, hipCimag(b));
}

__global__ void zeroOutState(int* new_state, long long int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) {
        new_state[idx] = 0;
    }
}


__global__ void updateState(Complex* state, Complex* new_state, long long int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) {
        state[idx] = new_state[idx];
    }
}


// __global__ void compute_idx(
//         int qubit,
//         const int* shape,
//         int* new_idx,
//         int* old_idx,
//         const int n,
//         const long long int N,
//         int* old_linear_idxs
//     ) {
//     int idx = blockDim.x * blockIdx.x + threadIdx.x;
//     int offset = idx * n;
//     // int offset2 = blockDim.x * gridDim.x;
//     int offset2 = qubit*2*N;
//     // TODO: make shape shared
//     //       make offset2 coalesced
//     //       use reduction for old_linear_idx
//     //       run all idxs for all qubits in parallel
//     // printf("offset: %d\n", offset);
//     // printf("offset2: %d\n", offset2);

//     if (idx < N) {
//         int temp = idx;

//         // Compute the multi-dimensional index
//         for (int i = n - 1; i >= 0; --i) {
//             new_idx[offset + i] = temp % shape[i];
//             temp /= shape[i];
//         }

//         // Copy new_idx to old_idx
//         for (int i = 0; i < n; ++i) {
//             old_idx[offset + i] = new_idx[offset + i];
//         }

//         // Compute the two values for j = 0 and j = 1 and store in shared memory
//         for (int j = 0; j < 2; ++j) {
//             old_idx[offset + qubit] = j;

//             // Compute the linear index for old_idx
//             int old_linear_idx = 0;
//             int factor = 1;
//             for (int i = n - 1; i >= 0; --i) {
//                 old_linear_idx += old_idx[offset + i] * factor;
//                 factor *= shape[i];
//             }
//             // old_linear_idxs[idx + j*N] = old_linear_idx;
//             // printf("idx: %d, old_lin_idx_pos: %lld = %d \n", idx, (idx + j*N), old_linear_idx);
//             // printf("idx: %d, old_lin_idx_pos: %d \n", idx, (idx + j*N));
//             // old_linear_idxs[offset + offset2 * j + qubit] = old_linear_idx;
//             old_linear_idxs[idx + j*N + offset2] = old_linear_idx;

//             // old_linear_idxs[idx + j*N] = old_linear_idx;

//         }
//     }
// }

__global__ void compute_idx(
        int qubit,
        int* new_idx,
        int* old_idx,
        const int n,
        const long long int N,
        int* shared_idxs,
        const int warp_size
    ) {
    // extern __shared__ int shared_memory[]; // Use shared memory
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = idx * n;
    // int offset2 = blockDim.x * gridDim.x;
    int offset2 = qubit*2*warp_size*N;
    // TODO: make shape shared - yes
    //       make offset2 coalesced
    //       use reduction for old_linear_idx
    //       run all idxs for all qubits in parallel
    // printf("offset: %d\n", offset);
    // printf("offset2: %d\n", offset2);

    if (idx < N) {
        int temp = idx;

        // Compute the multi-dimensional index
        for (int i = n - 1; i >= 0; --i) {
            new_idx[offset + i] = temp % 2;
            temp /= 2;
        }

        // Copy new_idx to old_idx
        for (int i = 0; i < n; ++i) {
            old_idx[offset + i] = new_idx[offset + i];
        }

        // Compute the two values for j = 0 and j = 1 and store in shared memory
        for (int j = 0; j < 2; ++j) {
            old_idx[offset + qubit] = j;

            // Compute the linear index for old_idx
            int old_linear_idx = 0;
            int factor = 1;
            for (int i = n - 1; i >= 0; --i) {
                // old_linear_idx += old_idx[offset + i] * factor;

                // TODO: make shared_mem
                // printf("idx: %d, j: %d, old_idx[offset + i] * factor: %d, offset: %d\n", idx, j, old_idx[offset + i] * factor, idx*2*warp_size + i + j*warp_size + offset2);
                shared_idxs[idx*2*warp_size + i + j*warp_size + offset2] = old_idx[offset + i] * factor;
                factor *= 2;
            }
            // old_linear_idxs[idx + j*N] = old_linear_idx;
            // printf("idx: %d, old_lin_idx_pos: %lld = %d \n", idx, (idx + j*N), old_linear_idx);
            // printf("idx: %d, old_lin_idx_pos: %d \n", idx, (idx + j*N));
            // old_linear_idxs[offset + offset2 * j + qubit] = old_linear_idx;
            // old_linear_idxs[idx + j*N + offset2] = old_linear_idx;
            // __syncthreads();
            // int val = shared_memory[idx*32 + j*32*N];
            // // int val = shared_memory[idx*32];

            // for (int i = 16; i > 0; i /= 2) {
            //     val += __shfl_down_sync(-1, val, i);
            // }
            // old_linear_idxs[idx + j*N + offset2] = val;
            // old_linear_idxs[idx + j*N] = old_linear_idx;


        }
        // shared_idxs = shared_memory;
    }
}

__global__ void warp_sum_reduction(const int* input, int* output, int N, int warp_size) {
    // Assuming N is a multiple of the warp size
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int lane = tid % warp_size; // Get the lane index within the warp
    int warp_id = tid / warp_size; // Get the warp ID

    int val = 0;
    if (tid < N) {
        val = input[tid];
    }

    // Perform warp-level reduction using shuffle down
    for (int offset = warp_size / 2; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset, warp_size);
    }

    // Write the reduced value of each warp to the output array
    if (lane == 0 && warp_id < (N/warp_size)) {
        output[warp_id] = val;
    }
}


__global__ void contract_tensor(
        Complex* state,
        const Complex* gate,
        int qubit,
        int* new_idx,
        int* old_idx,
        const int n,
        const long long int N,
        int* old_linear_idxs
    ) {
    extern __shared__ Complex shared_mem[]; // Use shared memory
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = idx * n;
    int offset2 = qubit*2*N;

    if (idx < N) {
        // Compute the two values for j = 0 and j = 1 and store in shared memory
        for (int j = 0; j < 2; ++j) {
            // Store the result in shared memory
            shared_mem[2*idx + j] = hipCmul(gate[new_idx[offset + qubit] * 2 + j], state[old_linear_idxs[2*idx + j + qubit*2*N]]);
        }
        __syncthreads();

        state[idx] = hipCadd(shared_mem[2*idx + 1], shared_mem[2*idx]);
    }
}


__global__ void applyPhaseFlip(Complex* state, long long int idx) {
    state[idx] = hipCmul(state[idx], make_hipDoubleComplex(-1.0, 0.0));
}

void applyGateAllQubits(
    Complex* state,
    const Complex* gate,
    int* new_idx,
    int* old_idx,
    int n,
    long long int N,
    dim3 dimBlock,
    dim3 dimGrid,
    int sharedMemSize,
    int* old_linear_idxs
    ) {

    for (int i = 0; i < n; ++i) {
        contract_tensor<<<dimGrid, dimBlock, sharedMemSize>>>(state, gate, i, new_idx, old_idx, n, N, old_linear_idxs);
    }
}

void applyGateSingleQubit(
    Complex* state,
    const Complex* gate,
    int* new_idx,
    int* old_idx,
    int n,
    long long int N,
    long long int idx,
    dim3 dimBlock,
    dim3 dimGrid,
    int sharedMemSize,
    int* old_linear_idxs
    ) {

    contract_tensor<<<dimGrid, dimBlock, sharedMemSize>>>(state, gate, idx, new_idx, old_idx, n, N, old_linear_idxs);
}

void applyDiffusionOperator(
    Complex* state,
    const Complex* X_H,
    const Complex* H,
    const Complex* X,
    const Complex* Z,
    int* new_idx,
    int* old_idx,
    int n,
    long long int N,
    dim3 dimBlock,
    dim3 dimGrid,
    int sharedMemSize,
    int* old_linear_idxs
    ) {
    applyGateAllQubits(state, X_H, new_idx, old_idx, n, N, dimBlock, dimGrid, sharedMemSize, old_linear_idxs);
    applyPhaseFlip<<<dimGrid, dimBlock>>>(state, N - 1);
    applyGateSingleQubit(state, Z, new_idx, old_idx, n, N, 0, dimBlock, dimGrid, sharedMemSize, old_linear_idxs);
    applyGateAllQubits(state, X, new_idx, old_idx, n, N, dimBlock, dimGrid, sharedMemSize, old_linear_idxs);
    applyGateSingleQubit(state, Z, new_idx, old_idx, n, N, 0, dimBlock, dimGrid, sharedMemSize, old_linear_idxs);
    applyGateAllQubits(state, H, new_idx, old_idx, n, N, dimBlock, dimGrid, sharedMemSize, old_linear_idxs);
}

// double* simulate(const Complex* weights, int numElements, int numSamples) {
//     if (numElements <= 0 || numSamples <= 0) {
//         fprintf(stderr, "Invalid input parameters.\n");
//         return NULL;
//     }

//     // Array to count occurrences of each index
//     int* counts = (int*)calloc(numElements, sizeof(int));
//     // Array to store the average frequencies
//     double* averages = (double*)calloc(numElements, sizeof(double));

//     if (counts == NULL || averages == NULL) {
//         fprintf(stderr, "Memory allocation failed.\n");
//         free(counts);
//         free(averages);
//         return NULL;
//     }

//     // Prepare weights for the distribution by extracting their magnitudes
//     double* magnitudes = (double*)malloc(numElements * sizeof(double));
//     if (magnitudes == NULL) {
//         fprintf(stderr, "Memory allocation failed.\n");
//         free(counts);
//         free(averages);
//         return NULL;
//     }

//     for (int i = 0; i < numElements; ++i) {
//         magnitudes[i] = cabs(weights[i]);
//     }

//     // Simulate the weighted distribution
//     for (int i = 0; i < numSamples; ++i) {
//         double r = (double)rand() / RAND_MAX;
//         double cum_prob = 0.0;
//         for (int j = 0; j < numElements; ++j) {
//             cum_prob += magnitudes[j];
//             if (r < cum_prob) {
//                 counts[j]++;
//                 break;
//             }
//         }
//     }

//     for (int i = 0; i < numElements; ++i) {
//         averages[i] = (double)counts[i] / numSamples;
//     }

//     free(counts);
//     free(magnitudes);
//     return averages;
// }

// Complex** createMatrix(int numRows, int numCols, const Complex* initialValues) {
//     if (numRows <= 0 || numCols <= 0) {
//         fprintf(stderr, "Invalid matrix dimensions.\n");
//         return NULL;
//     }

//     // Allocate memory for row pointers
//     Complex** matrix = (Complex**)malloc(numRows * sizeof(Complex*));
//     if (matrix == NULL) {
//         fprintf(stderr, "Memory allocation failed.\n");
//         return NULL;
//     }

//     // Allocate memory for each row and initialize with provided values
//     for (int i = 0; i < numRows; ++i) {
//         matrix[i] = (Complex*)malloc(numCols * sizeof(Complex));
//         if (matrix[i] == NULL) {
//             for (int j = 0; j < i; ++j) {
//                 free(matrix[j]);
//             }
//             free(matrix);
//             fprintf(stderr, "Memory allocation failed.\n");
//             return NULL;
//         }
//         for (int j = 0; j < numCols; ++j) {
//             int index = i * numCols + j;
//             matrix[i][j] = initialValues[index];
//         }
//     }

//     return matrix;
// }

// void deleteMatrix(Complex** matrix, int rows) {
//     for (int i = 0; i < rows; ++i) {
//         free(matrix[i]);
//     }
//     free(matrix);
// }

// Complex** kroneckerProduct(Complex** A, int aRows, int aCols, Complex** B, int bRows, int bCols) {
//     int resultRows = aRows * bRows;
//     int resultCols = aCols * bCols;
//     Complex** result = (Complex**)malloc(resultRows * sizeof(Complex*));
//     for (int i = 0; i < resultRows; ++i) {
//         result[i] = (Complex*)malloc(resultCols * sizeof(Complex));
//     }

//     for (int i = 0; i < aRows; ++i) {
//         for (int j = 0; j < aCols; ++j) {
//             for (int k = 0; k < bRows; ++k) {
//                 for (int l = 0; l < bCols;) {
//                     result[i * bRows + k][j * bCols + l] = A[i][j] * B[k][l];
//                 }
//             }
//         }
//     }

//     return result;
// }

// void printMatrix(Complex** matrix, int rows, int cols) {
//     for (int i = 0; i < rows; ++i) {
//         for (int j = 0; j < cols; ++j) {
//             printf("(%f + %fi) ", creal(matrix[i][j]), cimag(matrix[i][j]));
//         }
//         printf("\n");
//     }
// }
