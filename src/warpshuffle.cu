
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void warp_sum_reduction(const int* input, int* output, int N, int warp_size) {
    // Assuming N is a multiple of 32 (the warp size)
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int lane = tid % warp_size; // Get the lane index within the warp
    int warp_id = tid / warp_size; // Get the warp ID

    int val = 0;
    if (tid < N) {
        val = input[tid];
    }

    // Perform warp-level reduction using shuffle down
    for (int offset = warp_size / 2; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset, warp_size);
    }

    // Write the reduced value of each warp to the output array
    if (lane == 0 && warp_id < (N/warp_size)) {
        output[warp_id] = val;
    }
}

int main() {
    int N = 512;
    int warp_size = 64;
    int out_size = N / warp_size;
    int *d_input, *d_output;
    int *h_input, *h_output;
    hipMalloc(&d_input, N * sizeof(int));
    hipMalloc(&d_output, out_size * sizeof(int));
    hipHostMalloc(&h_input, N * sizeof(int), hipHostMallocDefault);
    hipHostMalloc(&h_output, out_size * sizeof(int), hipHostMallocDefault);

    for (int i = 1; i < N; ++i) {
        h_input[i] = 1;
    }

    // Fill d_input with data (omitted here for brevity)

    int threads_per_block = 512;
    int blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;

    hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice);


    warp_sum_reduction<<<blocks_per_grid, threads_per_block>>>(d_input, d_output, N, warp_size);

    hipMemcpy(h_output, d_output, out_size * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < out_size; ++i) {
            printf("%d ", h_output[i]);
        }

    // Copy results from device to host (omitted here for brevity)

    hipFree(d_input);
    hipFree(d_output);
}








