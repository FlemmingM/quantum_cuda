#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <math.h>
#include <string.h>
#include <errno.h>
#include <omp.h>
#include "utils_cuda.h"
#include "utils_cuda_v3.h"


typedef hipDoubleComplex Complex;



int main(int argc, char* argv[]) {

    // collect input args
    // if (argc < 6) {
    //     fprintf(stderr, "Usage: %s n qubits<int>; marked state<int>; number of samples<int>; fileName<string>; verbose 0 or 1<int>\n", argv[0]);
    //     return 1;
    // }

    int n = atoi(argv[1]);
    long long int N = pow(2, n);
    long long int markedState = atoi(argv[2]);
    const int num_chunks_per_group = atoi(argv[3]);
    const int num_qubits_per_group = atoi(argv[4]);

    if (markedState > (N-1)) {
        fprintf(stderr, "You chose a markedState %d but the largest state possible is state %d", markedState, (N-1));
        return 1;
    }


    // Define the number of groups to do the parallel search with more than 10 qubits
    // while still using the fast shared memory

    long long int num_groups = N / pow(2, num_qubits_per_group);
    int num_qubits_per_chunk = num_qubits_per_group - (int)log2(num_chunks_per_group);
    long long int N_chunk = pow(2, num_qubits_per_chunk);
    long long int N_group = num_chunks_per_group * N_chunk;
    long long int num_chunks = num_groups * num_chunks_per_group;

    if (N_chunk > pow(2, 10)) {
        fprintf(stderr, "You chose a number of qubits per group of: %d and a number of chunks per group of: %d\n Change the config so that the number of qubits per chunk is maximally 10 to fit into 1 block", num_qubits_per_group, num_chunks_per_group);
        return 1;
    }

    int sharedMemSize = N_chunk * sizeof(Complex);


    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    if (sharedMemSize > deviceProp.sharedMemPerBlock) {
        printf("Requested shared memory size (%d bytes) exceeds the device limit (%d bytes).\n", sharedMemSize, deviceProp.sharedMemPerBlock);
        return -1;
    }


    long long int oracle_group = markedState / (N / num_groups);


    markedState = markedState % (N / num_groups);
    long long int recoveredState = oracle_group*(N / num_groups)+markedState;


    dim3 dimBlock(N_chunk);
    dim3 dimBlock2(2*N_chunk);
    dim3 dimGrid(num_chunks_per_group);


    int print_val = 1;
    if (print_val == 1) {
        printf("N: %lld\n", N);
        printf("n: %d\n", n);
        printf("num_groups: %lld\n", num_groups);
        printf("num_chunks_per_group: %d\n", num_chunks_per_group);
        printf("num_qubits_per_chunk: %d\n", num_qubits_per_chunk);
        printf("N_chunk: %lld\n", N_chunk);
        printf("N_group: %lld\n", N_group);
        printf("num_chunks: %lld\n", num_chunks);
        printf("oracle_group: %lld, pos: %lld, recovered: %lld\n", oracle_group, markedState, recoveredState);
        printf("dimGrid: %d, dimBlock: %d\n", dimGrid.x, dimBlock.x);
    }


    // Set the gates:
    int num_devices = 1;
    Complex *H_d[num_devices];
    Complex *I_d[num_devices];
    Complex *Z_d[num_devices];
    Complex *X_d[num_devices];
    Complex *X_H_d[num_devices];
    allocateGatesDevice(num_devices, H_d, I_d, Z_d, X_d, X_H_d);



    // // Assuming we have t = 1 solution in grover's algorithm
    // // we have k = floor(pi/4 * sqrt(N/num_chunks))
    long long int k = (int)floor(M_PI / 4 * sqrt(N/num_chunks));
    printf("running %lld rounds\n", k);



    double time = omp_get_wtime();

    Complex *state_h;
    Complex *state_d;
    int *new_idx_d;
    int *new_idx_h;
    int *old_idx_d;
    int *old_linear_idxs_d;
    int *old_linear_idxs_h;

    // for indices
    int sharedMemSize2 = 2*N_chunk * sizeof(int);

    hipHostMalloc(&old_linear_idxs_h, 2 * N_chunk * num_qubits_per_chunk * sizeof(int));
    hipMalloc(&old_linear_idxs_d, 2 * N_chunk * num_qubits_per_chunk * sizeof(int));


    // init the arrays:
    hipHostMalloc((void **)&state_h, N_group * sizeof(Complex));
    // for (int i = 0; i < N_group; ++i) {
    //     if ((i % N_chunk)==0) {
    //         state_h[i] = make_hipDoubleComplex(1.0, 0.0);
    //     } else {
    //         state_h[i] = make_hipDoubleComplex(0.0, 0.0);
    //     }
    // }

    hipMalloc((void **)&state_d, N_group * sizeof(Complex));
    hipMalloc(&new_idx_d, N_group * n * sizeof(int));
    hipMalloc(&old_idx_d, N_group * n * sizeof(int));
    // hipMemcpy(state_d, state_h, N_group * sizeof(Complex), hipMemcpyHostToDevice);
    hipHostMalloc(&new_idx_h, N_chunk * num_qubits_per_chunk * sizeof(int));



    for (int i = 0; i < num_qubits_per_chunk; ++i) {
        compute_idx<<<1, dimBlock, sharedMemSize2>>>(i, new_idx_d, old_idx_d, num_qubits_per_chunk, N_chunk, old_linear_idxs_d);
    }

    // hipMemcpy(old_linear_idxs_h, old_linear_idxs_d, 2*N_chunk* num_qubits_per_chunk * sizeof(int), hipMemcpyDeviceToHost);

    // for (int i = 0; i < (2*N_chunk*num_qubits_per_chunk); ++i) {
    //     printf("%d ", old_linear_idxs_h[i]);
    // }
    // printf("\n");

    // hipMemcpy(new_idx_h, new_idx_d, N_chunk * num_qubits_per_chunk * sizeof(int), hipMemcpyDeviceToHost);

    // for (int i = 0; i < (N_chunk * num_qubits_per_chunk); ++i) {
    //     printf("%d ", new_idx_h[i]);
    // }
    // printf("\n");

    // initStateParallel<<<dimGrid, dimBlock>>>(state_d, N_group, N_chunk);
    // contract_tensor<<<dimGrid, dimBlock, sharedMemSize>>>(state_d, H_d[0], 0, new_idx_d, n, N, old_linear_idxs_d);
    // contract_tensor<<<dimGrid, dimBlock, sharedMemSize>>>(state_d, H_d[0], 1, new_idx_d, n, N, old_linear_idxs_d);
    // contract_tensor<<<dimGrid, dimBlock, sharedMemSize>>>(state_d, H_d[0], 2, new_idx_d, n, N, old_linear_idxs_d);


    // applyGateAllQubits(
    //     state_d,
    //     H_d[0], new_idx_d,
    //     num_qubits_per_chunk,
    //     dimBlock,
    //     dimGrid,
    //     sharedMemSize,
    //     N_group,
    //     old_linear_idxs_d
    // );


    // hipMemcpy(state_h, state_d, N_group * sizeof(Complex), hipMemcpyDeviceToHost);
    // printState(state_h, N_group, "state end");

double time2 = omp_get_wtime();
// for (int i = 0; i < num_qubits_per_chunk; ++i) {
//         compute_idx<<<1, dimBlock, sharedMemSize2>>>(i, new_idx_d, old_idx_d, num_qubits_per_chunk, N_chunk, old_linear_idxs_d);
//     }

// initStateParallel<<<dimGrid, dimBlock>>>(state_d, N_group, N_chunk);
// contract_tensor<<<dimGrid, dimBlock2, 2*sharedMemSize>>>(state_d, H_d[0], 0, new_idx_d, num_qubits_per_chunk, 2*N_group, old_linear_idxs_d);
// contract_tensor<<<dimGrid, 16, 2*sharedMemSize>>>(state_d, H_d[0], 1, new_idx_d, num_qubits_per_chunk, 2*N_group, old_linear_idxs_d);
// contract_tensor<<<dimGrid, 16, 2*sharedMemSize>>>(state_d, H_d[0], 2, new_idx_d, num_qubits_per_chunk, 2*N_group, old_linear_idxs_d);


// applyGateAllQubits(
//         state_d,
//         H_d[0], new_idx_d,
//         num_qubits_per_chunk,
//         dimBlock,
//         dimGrid,
//         sharedMemSize,
//         N_group,
//         old_linear_idxs_d
//     );


for (int i = 0; i < num_groups; ++i) {
    // reset the state vector for the next group
    initStateParallel<<<dimGrid, dimBlock>>>(state_d, N_group, N_chunk);

    applyGateAllQubits(
        state_d,
        H_d[0], new_idx_d,
        num_qubits_per_chunk,
        dimBlock2,
        dimGrid,
        2*sharedMemSize,
        2*N_group,
        old_linear_idxs_d
    );

    // applyPhaseFlip<<<dimGrid, dimBlock, 0>>>(state_d, markedState);
    // printf("\n");
    // applyGateAllQubits(
    //     state_d,
    //     X_H_d[0], new_idx_d,
    //     num_qubits_per_chunk,
    //     dimBlock2,
    //     dimGrid,
    //     2*sharedMemSize,
    //     2*N_group,
    //     old_linear_idxs_d
    // );

    for (int l = 0; l < k; ++l) {
        if (i == oracle_group) {
            // printf("oracle chunk_id: %lld, i: %d\n", oracle_group, i);
            applyPhaseFlip<<<dimGrid, dimBlock2, 0>>>(state_d, markedState);
        }

        applyDiffusionOperator(
            state_d,
            X_H_d[0], H_d[0], X_d[0], Z_d[0], new_idx_d,
            num_qubits_per_chunk, dimBlock2, dimGrid, 2*sharedMemSize,
            num_chunks_per_group,
            N_chunk,
            2*N_group,
            old_linear_idxs_d
        );
    }

    if (i == oracle_group) {
        hipMemcpy(state_h, state_d, N_group * sizeof(Complex), hipMemcpyDeviceToHost);
        // printState(state_h, N_group, "state end");
    }
    hipDeviceSynchronize();
}

    // hipMemcpy(state_h, state_d, N_group * sizeof(Complex), hipMemcpyDeviceToHost);
    // printState(state_h, N_group, "state end");

    double elapsed2 = omp_get_wtime() - time2;
    printf("Time compute: %f \n", elapsed2);
    double elapsed = omp_get_wtime() - time;
    printf("Time: %f \n", elapsed);
    // // n, k, num_groups, num_chunks, n_per_group, chunks_per_group, num_threads, marked_chunk, markedState, marked_max_idx, marked_max_val, time
    printf("%d,%lld,%lld,%lld,%d,%d,%d,%d,%f\n",
        n, k, num_groups, num_chunks, num_qubits_per_group, num_chunks_per_group, dimBlock.x, markedState, elapsed);


    hipFree(H_d[0]);
    hipFree(I_d[0]);
    hipFree(Z_d[0]);
    hipFree(X_d[0]);
    hipFree(X_H_d[0]);

    hipFree(new_idx_d);
    hipFree(old_idx_d);
    hipFree(state_d);
    hipHostFree(state_h);

    return 0;
}
