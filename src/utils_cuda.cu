
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <math.h>
#include <string.h>
#include <errno.h>
#include <omp.h>

typedef hipDoubleComplex Complex;

void saveArrayToCSV(const double *array, long long int N, const char* filename) {
    FILE *file = fopen(filename, "w");

    if (!file) {
        perror("Unable to open file");
        return;
    }
    fprintf(file, "position,probability\n");
    for (int i = 0; i < N; ++i) {
        fprintf(file, "pos%d,%f\n", i, array[i]);
    }
    fclose(file);
}


__device__ void AddComplex(hipDoubleComplex* a, hipDoubleComplex b){
  //transform the addresses of real and imag. parts to double pointers
  double *x = (double*)a;
  double *y = x+1;
  //use atomicAdd for double variables
  atomicAdd(x, hipCreal(b));
  atomicAdd(y, hipCimag(b));
}

__global__ void zeroOutState(Complex* new_state, long long int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) {
        new_state[idx] = make_hipDoubleComplex(0.0, 0.0);
    }
}


__global__ void updateState(Complex* state, Complex* new_state, long long int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) {
        state[idx] = new_state[idx];
    }
}

__global__ void contract_tensor_baseline(
        const Complex* state,
        const Complex* gate,
        int qubit,
        Complex* new_state,
        const int* shape,
        int* new_idx,
        int* old_idx,
        const int n,
        const long long int N
    ) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    // int offset = blockDim.x * gridDim.x;
    int offset = idx * n;
    if (idx < N) {

        int temp = idx;

        // Compute the multi-dimensional index
        for (int i = n - 1; i >= 0; --i) {
            new_idx[offset+i] = temp % shape[i];
            temp /= shape[i];
        }

        // Perform the tensor contraction for the specified qubit
        for (int j = 0; j < 2; ++j) {
            // Copy new_idx to old_idx
            for (int i = 0; i < n; ++i) {
                old_idx[offset+i] = new_idx[offset+i];
            }
            old_idx[offset+qubit] = j;

            // Compute the linear index for old_idx
            int old_linear_idx = 0;
            int factor = 1;
            for (int i = n - 1; i >= 0; --i) {
                old_linear_idx += old_idx[offset+i] * factor;
                factor *= shape[i];
            }
            AddComplex(&new_state[idx], hipCmul(gate[new_idx[offset+qubit] * 2 + j], state[old_linear_idx]));
            }
        }
    }

void printState(const Complex* state, long long int N, const char* message) {
    printf("%s\n", message);
    for (int i = 0; i < N; ++i) {
        printf("(%.15f + %.15fi) ", hipCreal(state[i]), hipCimag(state[i]));
    }
    printf("\n");
}

__global__ void applyPhaseFlip(Complex* state, long long int idx) {
    state[idx] = hipCmul(state[idx], make_hipDoubleComplex(-1.0, 0.0));
}

void applyGateAllQubits(
    Complex* state,
    const Complex* gate,
    Complex* new_state,
    const int* shape,
    int* new_idx,
    int* old_idx,
    int n,
    long long int N,
    dim3 dimBlock,
    dim3 dimGrid
    ) {

    for (int i = 0; i < n; ++i) {
        contract_tensor_baseline<<<dimGrid, dimBlock>>>(state, gate, i, new_state, shape, new_idx, old_idx, n, N);
        // contract_tensor_baseline<<<dimGrid, dimBlock>>>(state, gate, i, new_state, shape, n, N);
        hipDeviceSynchronize();
        // Update the state with the new state
        updateState<<<dimGrid, dimBlock>>>(state, new_state, N);
        hipDeviceSynchronize();
        zeroOutState<<<dimGrid, dimBlock>>>(new_state, N);
        hipDeviceSynchronize();
    }
}

// void applyGateSingleQubit(
//     Complex* state,
//     const Complex* gate,
//     Complex* new_state,
//     const int* shape,
//     int n,
//     long long int N,
//     long long int idx,
//     dim3 dimBlock,
//     dim3 dimGrid
//     ) {

//     contract_tensor_baseline<<<dimGrid, dimBlock>>>(state, gate, idx, new_state, shape, n, N);
//     // Update the state with the new state
//     updateState<<<dimGrid, dimBlock>>>(state, new_state, N);
//     zeroOutState<<<dimGrid, dimBlock>>>(new_state, N);
// }

// void applyDiffusionOperator(
//     Complex* state,
//     Complex* new_state,
//     const int* shape,
//     const Complex* H,
//     const Complex* X,
//     const Complex* Z,
//     int n,
//     long long int N,
//     dim3 dimBlock,
//     dim3 dimGrid
//     ) {
//     applyGateAllQubits(state, H, new_state, shape, n, N, dimBlock, dimGrid);
//     applyGateAllQubits(state, X, new_state, shape, n, N, dimBlock, dimGrid);
//     applyPhaseFlip<<<dimGrid, dimBlock>>>(state, N - 1);
//     applyGateSingleQubit(state, Z, new_state, shape, n, N, 0, dimBlock, dimGrid);
//     applyGateAllQubits(state, X, new_state, shape, n, N, dimBlock, dimGrid);
//     applyGateSingleQubit(state, Z, new_state, shape, n, N, 0, dimBlock, dimGrid);
//     applyGateAllQubits(state, H, new_state, shape, n, N, dimBlock, dimGrid);
// }

// double* simulate(const Complex* weights, int numElements, int numSamples) {
//     if (numElements <= 0 || numSamples <= 0) {
//         fprintf(stderr, "Invalid input parameters.\n");
//         return NULL;
//     }

//     // Array to count occurrences of each index
//     int* counts = (int*)calloc(numElements, sizeof(int));
//     // Array to store the average frequencies
//     double* averages = (double*)calloc(numElements, sizeof(double));

//     if (counts == NULL || averages == NULL) {
//         fprintf(stderr, "Memory allocation failed.\n");
//         free(counts);
//         free(averages);
//         return NULL;
//     }

//     // Prepare weights for the distribution by extracting their magnitudes
//     double* magnitudes = (double*)malloc(numElements * sizeof(double));
//     if (magnitudes == NULL) {
//         fprintf(stderr, "Memory allocation failed.\n");
//         free(counts);
//         free(averages);
//         return NULL;
//     }

//     for (int i = 0; i < numElements; ++i) {
//         magnitudes[i] = cabs(weights[i]);
//     }

//     // Simulate the weighted distribution
//     for (int i = 0; i < numSamples; ++i) {
//         double r = (double)rand() / RAND_MAX;
//         double cum_prob = 0.0;
//         for (int j = 0; j < numElements; ++j) {
//             cum_prob += magnitudes[j];
//             if (r < cum_prob) {
//                 counts[j]++;
//                 break;
//             }
//         }
//     }

//     for (int i = 0; i < numElements; ++i) {
//         averages[i] = (double)counts[i] / numSamples;
//     }

//     free(counts);
//     free(magnitudes);
//     return averages;
// }

// Complex** createMatrix(int numRows, int numCols, const Complex* initialValues) {
//     if (numRows <= 0 || numCols <= 0) {
//         fprintf(stderr, "Invalid matrix dimensions.\n");
//         return NULL;
//     }

//     // Allocate memory for row pointers
//     Complex** matrix = (Complex**)malloc(numRows * sizeof(Complex*));
//     if (matrix == NULL) {
//         fprintf(stderr, "Memory allocation failed.\n");
//         return NULL;
//     }

//     // Allocate memory for each row and initialize with provided values
//     for (int i = 0; i < numRows; ++i) {
//         matrix[i] = (Complex*)malloc(numCols * sizeof(Complex));
//         if (matrix[i] == NULL) {
//             for (int j = 0; j < i; ++j) {
//                 free(matrix[j]);
//             }
//             free(matrix);
//             fprintf(stderr, "Memory allocation failed.\n");
//             return NULL;
//         }
//         for (int j = 0; j < numCols; ++j) {
//             int index = i * numCols + j;
//             matrix[i][j] = initialValues[index];
//         }
//     }

//     return matrix;
// }

// void deleteMatrix(Complex** matrix, int rows) {
//     for (int i = 0; i < rows; ++i) {
//         free(matrix[i]);
//     }
//     free(matrix);
// }

// Complex** kroneckerProduct(Complex** A, int aRows, int aCols, Complex** B, int bRows, int bCols) {
//     int resultRows = aRows * bRows;
//     int resultCols = aCols * bCols;
//     Complex** result = (Complex**)malloc(resultRows * sizeof(Complex*));
//     for (int i = 0; i < resultRows; ++i) {
//         result[i] = (Complex*)malloc(resultCols * sizeof(Complex));
//     }

//     for (int i = 0; i < aRows; ++i) {
//         for (int j = 0; j < aCols; ++j) {
//             for (int k = 0; k < bRows; ++k) {
//                 for (int l = 0; l < bCols;) {
//                     result[i * bRows + k][j * bCols + l] = A[i][j] * B[k][l];
//                 }
//             }
//         }
//     }

//     return result;
// }

// void printMatrix(Complex** matrix, int rows, int cols) {
//     for (int i = 0; i < rows; ++i) {
//         for (int j = 0; j < cols; ++j) {
//             printf("(%f + %fi) ", creal(matrix[i][j]), cimag(matrix[i][j]));
//         }
//         printf("\n");
//     }
// }
