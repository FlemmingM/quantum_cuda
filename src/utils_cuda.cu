
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <math.h>
#include <string.h>
#include <errno.h>
#include <omp.h>

typedef hipDoubleComplex Complex;

void saveArrayToCSV(const double *array, long long int N, const char* filename) {
    FILE *file = fopen(filename, "w");

    if (!file) {
        perror("Unable to open file");
        return;
    }
    fprintf(file, "position,probability\n");
    for (int i = 0; i < N; ++i) {
        fprintf(file, "pos%d,%f\n", i, array[i]);
    }
    fclose(file);
}

__global__ void initState(Complex* new_state, int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) {
        if (idx==0) {
            new_state[idx] = make_hipDoubleComplex(1.0, 0.0);
        } else {
            new_state[idx] = make_hipDoubleComplex(0.0, 0.0);
        }

    }
}


__device__ void AddComplex(hipDoubleComplex* a, hipDoubleComplex b){
  //transform the addresses of real and imag. parts to double pointers
  double *x = (double*)a;
  double *y = x+1;
  //use atomicAdd for double variables
  atomicAdd(x, hipCreal(b));
  atomicAdd(y, hipCimag(b));
}

__global__ void zeroOutState(Complex* new_state, long long int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) {
        new_state[idx] = make_hipDoubleComplex(0.0, 0.0);
    }
}


__global__ void updateState(Complex* state, Complex* new_state, long long int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) {
        state[idx] = new_state[idx];
    }
}


void printState(const Complex* state, long long int N, const char* message) {
    printf("%s\n", message);
    for (int i = 0; i < N; ++i) {
        printf("(%.15f + %.15fi) ", hipCreal(state[i]), hipCimag(state[i]));
    }
    printf("\n");
}

// double* simulate(const Complex* weights, int numElements, int numSamples) {
//     if (numElements <= 0 || numSamples <= 0) {
//         fprintf(stderr, "Invalid input parameters.\n");
//         return NULL;
//     }

//     // Array to count occurrences of each index
//     int* counts = (int*)calloc(numElements, sizeof(int));
//     // Array to store the average frequencies
//     double* averages = (double*)calloc(numElements, sizeof(double));

//     if (counts == NULL || averages == NULL) {
//         fprintf(stderr, "Memory allocation failed.\n");
//         free(counts);
//         free(averages);
//         return NULL;
//     }

//     // Prepare weights for the distribution by extracting their magnitudes
//     double* magnitudes = (double*)malloc(numElements * sizeof(double));
//     if (magnitudes == NULL) {
//         fprintf(stderr, "Memory allocation failed.\n");
//         free(counts);
//         free(averages);
//         return NULL;
//     }

//     for (int i = 0; i < numElements; ++i) {
//         magnitudes[i] = cabs(weights[i]);
//     }

//     // Simulate the weighted distribution
//     for (int i = 0; i < numSamples; ++i) {
//         double r = (double)rand() / RAND_MAX;
//         double cum_prob = 0.0;
//         for (int j = 0; j < numElements; ++j) {
//             cum_prob += magnitudes[j];
//             if (r < cum_prob) {
//                 counts[j]++;
//                 break;
//             }
//         }
//     }

//     for (int i = 0; i < numElements; ++i) {
//         averages[i] = (double)counts[i] / numSamples;
//     }

//     free(counts);
//     free(magnitudes);
//     return averages;
// }

// Complex** createMatrix(int numRows, int numCols, const Complex* initialValues) {
//     if (numRows <= 0 || numCols <= 0) {
//         fprintf(stderr, "Invalid matrix dimensions.\n");
//         return NULL;
//     }

//     // Allocate memory for row pointers
//     Complex** matrix = (Complex**)malloc(numRows * sizeof(Complex*));
//     if (matrix == NULL) {
//         fprintf(stderr, "Memory allocation failed.\n");
//         return NULL;
//     }

//     // Allocate memory for each row and initialize with provided values
//     for (int i = 0; i < numRows; ++i) {
//         matrix[i] = (Complex*)malloc(numCols * sizeof(Complex));
//         if (matrix[i] == NULL) {
//             for (int j = 0; j < i; ++j) {
//                 free(matrix[j]);
//             }
//             free(matrix);
//             fprintf(stderr, "Memory allocation failed.\n");
//             return NULL;
//         }
//         for (int j = 0; j < numCols; ++j) {
//             int index = i * numCols + j;
//             matrix[i][j] = initialValues[index];
//         }
//     }

//     return matrix;
// }

// void deleteMatrix(Complex** matrix, int rows) {
//     for (int i = 0; i < rows; ++i) {
//         free(matrix[i]);
//     }
//     free(matrix);
// }

// Complex** kroneckerProduct(Complex** A, int aRows, int aCols, Complex** B, int bRows, int bCols) {
//     int resultRows = aRows * bRows;
//     int resultCols = aCols * bCols;
//     Complex** result = (Complex**)malloc(resultRows * sizeof(Complex*));
//     for (int i = 0; i < resultRows; ++i) {
//         result[i] = (Complex*)malloc(resultCols * sizeof(Complex));
//     }

//     for (int i = 0; i < aRows; ++i) {
//         for (int j = 0; j < aCols; ++j) {
//             for (int k = 0; k < bRows; ++k) {
//                 for (int l = 0; l < bCols;) {
//                     result[i * bRows + k][j * bCols + l] = A[i][j] * B[k][l];
//                 }
//             }
//         }
//     }

//     return result;
// }

// void printMatrix(Complex** matrix, int rows, int cols) {
//     for (int i = 0; i < rows; ++i) {
//         for (int j = 0; j < cols; ++j) {
//             printf("(%f + %fi) ", creal(matrix[i][j]), cimag(matrix[i][j]));
//         }
//         printf("\n");
//     }
// }
