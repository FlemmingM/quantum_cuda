
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <math.h>
#include <string.h>
#include <errno.h>
#include <omp.h>

typedef hipDoubleComplex Complex;


void allocateGatesDevice(const int num_devices, Complex **H_d, Complex **I_d, Complex **Z_d, Complex **X_d, Complex **X_H_d) {

    // Define the gates
    hipDoubleComplex H_h[4] = {
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0),
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(-1.0 / sqrt(2.0), 0.0)
    };
    hipDoubleComplex X_H_h[4] = {
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(-1.0 / sqrt(2.0), 0.0),
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0)
    };
    hipDoubleComplex I_h[4] = {
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0),
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(1.0, 0.0)
    };
    hipDoubleComplex Z_h[4] = {
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0),
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(-1.0, 0.0)
    };
    hipDoubleComplex X_h[4] = {
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(1.0, 0.0),
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0)
    };

    for (int i = 0; i < num_devices; i++) {
        // Set the device
        hipSetDevice(i);

        // Malloc the gate on device
        hipMalloc((void **)&H_d[i], 4 * sizeof(Complex));
        hipMalloc((void **)&X_H_d[i], 4 * sizeof(Complex));
        hipMalloc((void **)&I_d[i], 4 * sizeof(Complex));
        hipMalloc((void **)&Z_d[i], 4 * sizeof(Complex));
        hipMalloc((void **)&X_d[i], 4 * sizeof(Complex));

        // Copy from host to device
        hipMemcpy(H_d[i], H_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
        hipMemcpy(X_H_d[i], X_H_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
        hipMemcpy(I_d[i], I_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
        hipMemcpy(Z_d[i], Z_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
        hipMemcpy(X_d[i], X_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
    }
}


__global__ void compute_idx(
        int qubit,
        int* new_idx,
        int* old_idx,
        const int n,
        const long long int N,
        int* old_linear_idxs
    ) {
    extern __shared__ int shared_memory[]; // Use shared memory

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = idx * n;
    // int chunk_size = pow(2, n);
    int offset2 = qubit*2*N;


    if (idx < N) {
        int temp = idx;

        // Compute the multi-dimensional index
        for (int i = n - 1; i >= 0; --i) {
            new_idx[offset + i] = temp % 2;
            temp /= 2;
        }

        // Copy new_idx to old_idx
        for (int i = 0; i < n; ++i) {
            old_idx[offset + i] = new_idx[offset + i];
        }

        // Compute the two values for j = 0 and j = 1 and store in shared memory
        for (int j = 0; j < 2; ++j) {
            old_idx[offset + qubit] = j;

            // Compute the linear index for old_idx
            int old_linear_idx = 0;
            int factor = 1;
            for (int i = n - 1; i >= 0; --i) {
                old_linear_idx += old_idx[offset + i] * factor;
                factor *= 2;
            }
            // coalesced
            shared_memory[2*idx + j] = old_linear_idx;
        }
        // coalesced
        old_linear_idxs[2*idx + offset2] = shared_memory[2*idx];
        old_linear_idxs[2*idx + 1 + offset2] = shared_memory[2*idx+1];
    }
}

__global__ void contract_tensor(
        Complex* state,
        const Complex* gate,
        int qubit,
        int* new_idx,
        const int n,
        const long long int N,
        int* old_linear_idxs

) {
    extern __shared__ Complex shared_mem[]; // Use shared memory
    int idx = blockDim.x * blockIdx.x + threadIdx.x;


    int chunk_size = pow(2, n);

    if (idx < N) {

        int offset = (idx % chunk_size) * n;

        // Compute the two values for j = 0 and j = 1 and store in shared memory
        for (int j = 0; j < 2; ++j) {
            // needed to translate back to the full state array!!!
            int old_linear_idx = old_linear_idxs[2*(idx % chunk_size) + j + qubit*2*chunk_size];
            old_linear_idx += (idx / chunk_size) * chunk_size;

            // Store the result in shared memory
            if (j == 0) {
                Complex val = hipCmul(gate[new_idx[offset + qubit] * 2 + j], state[old_linear_idx]);
                shared_mem[idx % chunk_size] = val;

            } else {
                Complex val = hipCmul(gate[new_idx[offset + qubit] * 2 + j], state[old_linear_idx]);
                shared_mem[idx % chunk_size] = hipCadd(shared_mem[idx % chunk_size], val);
            }

        }
        state[idx] = shared_mem[idx % chunk_size];
    }
}


__global__ void applyPhaseFlip(Complex* state, long long int idx) {
    state[idx] = hipCmul(state[idx], make_hipDoubleComplex(-1.0, 0.0));
}

void applyGateAllQubits(
    Complex* state,
    const Complex* gate,
    int* new_idx,
    int n,
    dim3 dimBlock,
    dim3 dimGrid,
    int sharedMemSize,
    const long long int N,
    int* old_linear_idxs
    ) {

    for (int i = 0; i < n; ++i) {
        contract_tensor<<<dimGrid, dimBlock, sharedMemSize>>>(state, gate, i, new_idx, n, N, old_linear_idxs);
    }
}

void applyGateSingleQubit(
    Complex* state,
    const Complex* gate,
    int* new_idx,
    int n,
    long long int idx,
    dim3 dimBlock,
    dim3 dimGrid,
    int sharedMemSize,
    const long long int N,
    int* old_linear_idxs
    ) {

    contract_tensor<<<dimGrid, dimBlock, sharedMemSize>>>(state, gate, idx, new_idx, n, N, old_linear_idxs);
}

void applyDiffusionOperator(
    Complex* state,
    const Complex* X_H,
    const Complex* H,
    const Complex* X,
    const Complex* Z,
    int* new_idx,
    int n,
    dim3 dimBlock,
    dim3 dimGrid,
    int sharedMemSize,
    const long long int num_chunks_per_group,
    const long long int N_chunk,
    const long long int N,
    int* old_linear_idxs
    ) {
    applyGateAllQubits(state, X_H, new_idx, n, dimBlock, dimGrid, sharedMemSize, N, old_linear_idxs);
    for (int i = 0; i < num_chunks_per_group; ++i) {
        applyPhaseFlip<<<dimGrid, dimBlock, 0>>>(state, (i+1)*N_chunk - 1);
    }

    applyGateSingleQubit(state, Z, new_idx, n, 0, dimBlock, dimGrid, sharedMemSize, N, old_linear_idxs);
    applyGateAllQubits(state, X, new_idx, n, dimBlock, dimGrid, sharedMemSize, N, old_linear_idxs);
    applyGateSingleQubit(state, Z, new_idx, n, 0, dimBlock, dimGrid, sharedMemSize, N, old_linear_idxs);
    applyGateAllQubits(state, H, new_idx, n, dimBlock, dimGrid, sharedMemSize, N, old_linear_idxs);
}
