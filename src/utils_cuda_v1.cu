
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <math.h>
#include <string.h>
#include <errno.h>
#include <omp.h>

typedef hipDoubleComplex Complex;


void allocateGatesDevice(const int num_devices, Complex **H_d, Complex **I_d, Complex **Z_d, Complex **X_d, Complex **X_H_d) {

    // Define the gates
    hipDoubleComplex H_h[4] = {
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0),
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(-1.0 / sqrt(2.0), 0.0)
    };
    hipDoubleComplex X_H_h[4] = {
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(-1.0 / sqrt(2.0), 0.0),
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0)
    };
    hipDoubleComplex I_h[4] = {
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0),
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(1.0, 0.0)
    };
    hipDoubleComplex Z_h[4] = {
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0),
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(-1.0, 0.0)
    };
    hipDoubleComplex X_h[4] = {
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(1.0, 0.0),
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0)
    };

    for (int i = 0; i < num_devices; i++) {
        // Set the device
        hipSetDevice(i);

        // Malloc the gate on device
        hipMalloc((void **)&H_d[i], 4 * sizeof(Complex));
        hipMalloc((void **)&X_H_d[i], 4 * sizeof(Complex));
        hipMalloc((void **)&I_d[i], 4 * sizeof(Complex));
        hipMalloc((void **)&Z_d[i], 4 * sizeof(Complex));
        hipMalloc((void **)&X_d[i], 4 * sizeof(Complex));

        // Copy from host to device
        hipMemcpy(H_d[i], H_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
        hipMemcpy(X_H_d[i], X_H_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
        hipMemcpy(I_d[i], I_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
        hipMemcpy(Z_d[i], Z_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
        hipMemcpy(X_d[i], X_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
    }
}


__global__ void findMaxIndexKernel(Complex* d_array, int* d_maxIndex, double* d_maxValue, int size, int chunk_id, int* chunk_ids) {
    __shared__ Complex sharedArray[1024];
    __shared__ int sharedIndex[1024];

    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size) {
        sharedArray[tid] = d_array[index];
        sharedIndex[tid] = index;
    } else {
        sharedArray[tid] = make_hipDoubleComplex(-99.0, 0.00);  // Set to minimum value if out of bounds
        sharedIndex[tid] = -1;        // Invalid index
    }

    __syncthreads();

    // Perform reduction to find the max value and its index
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride && index + stride < size) {
            if (hipCreal(sharedArray[tid]) < hipCreal(sharedArray[tid + stride])) {
                sharedArray[tid] = sharedArray[tid + stride];
                sharedIndex[tid] = sharedIndex[tid + stride];
            }
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tid == 0) {


        // printf("Val: %f, Index: %d, chunk_id: %d\n", cuCreal(sharedArray[0]), sharedIndex[0], chunk_id);
        // printf("Index: %d\n", sharedIndex[0]);
        // printf("chunk_id: %d\n", chunk_id);
        d_maxIndex[chunk_id] = sharedIndex[0];
        d_maxValue[chunk_id] = hipCreal(sharedArray[0]);
        chunk_ids[chunk_id] = chunk_id;
        // printf("Val: %f, Index: %d, chunk_id: %d\n", cuCreal(sharedArray[0]), sharedIndex[0], chunk_id);

        // for (int i = 0; i < 2; ++i){
        //     printf("d_maxIndex: %d\n", d_maxIndex[i]);
        // }

    }
}

__global__ void contract_tensor(
        Complex* state,
        const Complex* gate,
        int qubit,
        int* new_idx,
        int* old_idx,
        const int n,
        // const long long int N,
        const long long int lower,
        const long long int upper
) {
    extern __shared__ Complex shared_mem[]; // Use shared memory
    int idx = blockDim.x * blockIdx.x + threadIdx.x;


    // int chunk_size = upper-lower;
    int chunk_size = pow(2, n);

    if ((idx >= lower) & (idx < upper)) {

        int offset = idx * n;
        int temp = idx % chunk_size;
        // int temp = idx;

        // printf("idx: %d, temp: %d, offset: %d, lower %lld, upper %lld\n", idx, temp, offset, lower, upper);

        // Compute the multi-dimensional index
        for (int i = n - 1; i >= 0; --i) {
            new_idx[offset + i] = temp % 2;
            temp /= 2;
        }

        // Copy new_idx to old_idx
        for (int i = 0; i < n; ++i) {
            old_idx[offset + i] = new_idx[offset + i];
        }

        // Compute the two values for j = 0 and j = 1 and store in shared memory
        for (int j = 0; j < 2; ++j) {
            old_idx[offset + qubit] = j;

            // Compute the linear index for old_idx
            int old_linear_idx = 0;
            int factor = 1;
            for (int i = n - 1; i >= 0; --i) {
                old_linear_idx += old_idx[offset + i] * factor;
                factor *= 2;
            }

            // needed to translate back to the full state array!!!
            if (idx >= chunk_size) {
                old_linear_idx += (idx / chunk_size) * chunk_size;
            }

            // Store the result in shared memory
            if (j == 0) {
                Complex val = hipCmul(gate[new_idx[offset + qubit] * 2 + j], state[old_linear_idx]);
                shared_mem[idx % chunk_size] = val;
                printf("idx: %d, j: %d, old_lin_idx %d, val: %f\n", idx, j, old_linear_idx, hipCreal(val));

            } else {
                Complex val = hipCmul(gate[new_idx[offset + qubit] * 2 + j], state[old_linear_idx]);
                shared_mem[idx % chunk_size] = hipCadd(shared_mem[idx % chunk_size], val);
                // printf("idx: %d, j: %d, old_lin_idx %d, val: %f\n", idx, j, old_linear_idx, cuCreal(val));
            }
            // printf("idx: %d, temp: %d, offset: %d, old_lin_idx %d, upper %lld\n", idx, temp, offset, old_linear_idx, upper);

        }
        __syncthreads();
        state[idx] = shared_mem[idx % chunk_size];
    }
}


__global__ void applyPhaseFlip(Complex* state, long long int idx) {
    state[idx] = hipCmul(state[idx], make_hipDoubleComplex(-1.0, 0.0));
}

void applyGateAllQubits(
    Complex* state,
    const Complex* gate,
    int* new_idx,
    int* old_idx,
    int n,
    dim3 dimBlock,
    dim3 dimGrid,
    int sharedMemSize,
    const long long int lower,
    const long long int upper
    ) {

    for (int i = 0; i < n; ++i) {
        contract_tensor<<<dimGrid, dimBlock, sharedMemSize>>>(state, gate, i, new_idx, old_idx, n, lower, upper);
    }
}

void applyGateSingleQubit(
    Complex* state,
    const Complex* gate,
    int* new_idx,
    int* old_idx,
    int n,
    long long int idx,
    dim3 dimBlock,
    dim3 dimGrid,
    int sharedMemSize,
    const long long int lower,
    const long long int upper
    ) {

    contract_tensor<<<dimGrid, dimBlock, sharedMemSize>>>(state, gate, idx, new_idx, old_idx, n, lower, upper);
}

void applyDiffusionOperator(
    Complex* state,
    const Complex* X_H,
    const Complex* H,
    const Complex* X,
    const Complex* Z,
    int* new_idx,
    int* old_idx,
    int n,
    dim3 dimBlock,
    dim3 dimGrid,
    int sharedMemSize,
    const int num_chunks_per_group,
    const long long int N_chunk,
    const long long int lower,
    const long long int upper
    ) {
    applyGateAllQubits(state, X_H, new_idx, old_idx, n, dimBlock, dimGrid, sharedMemSize, lower, upper);
    for (int i = 0; i < num_chunks_per_group; ++i) {
        applyPhaseFlip<<<dimGrid, dimBlock, 0>>>(state, (i+1)*N_chunk - 1);
        // applyGateSingleQubit(state, Z, new_idx, old_idx, n, i*N_chunk, dimBlock, dimGrid, sharedMemSize, lower, upper);
    }

    applyGateSingleQubit(state, Z, new_idx, old_idx, n, 0, dimBlock, dimGrid, sharedMemSize, lower, upper);
    applyGateAllQubits(state, X, new_idx, old_idx, n, dimBlock, dimGrid, sharedMemSize, lower, upper);
    // for (int i = 0; i < num_chunks_per_group; ++i) {
    //     applyGateSingleQubit(state, Z, new_idx, old_idx, n, i*N_chunk, dimBlock, dimGrid, sharedMemSize, lower, upper);
    // }
    applyGateSingleQubit(state, Z, new_idx, old_idx, n, 0, dimBlock, dimGrid, sharedMemSize, lower, upper);
    applyGateAllQubits(state, H, new_idx, old_idx, n, dimBlock, dimGrid, sharedMemSize, lower, upper);
}
