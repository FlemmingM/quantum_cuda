
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <math.h>
#include <string.h>
#include <errno.h>
#include <omp.h>

typedef hipDoubleComplex Complex;


void allocateGatesDevice(const int num_devices, Complex **H_d, Complex **I_d, Complex **Z_d, Complex **X_d, Complex **X_H_d) {

    // Define the gates
    hipDoubleComplex H_h[4] = {
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0),
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(-1.0 / sqrt(2.0), 0.0)
    };
    hipDoubleComplex X_H_h[4] = {
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(-1.0 / sqrt(2.0), 0.0),
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0)
    };
    hipDoubleComplex I_h[4] = {
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0),
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(1.0, 0.0)
    };
    hipDoubleComplex Z_h[4] = {
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0),
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(-1.0, 0.0)
    };
    hipDoubleComplex X_h[4] = {
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(1.0, 0.0),
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0)
    };

    for (int i = 0; i < num_devices; i++) {
        // Set the device
        hipSetDevice(i);

        // Malloc the gate on device
        hipMalloc((void **)&H_d[i], 4 * sizeof(Complex));
        hipMalloc((void **)&X_H_d[i], 4 * sizeof(Complex));
        hipMalloc((void **)&I_d[i], 4 * sizeof(Complex));
        hipMalloc((void **)&Z_d[i], 4 * sizeof(Complex));
        hipMalloc((void **)&X_d[i], 4 * sizeof(Complex));

        // Copy from host to device
        hipMemcpy(H_d[i], H_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
        hipMemcpy(X_H_d[i], X_H_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
        hipMemcpy(I_d[i], I_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
        hipMemcpy(Z_d[i], Z_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
        hipMemcpy(X_d[i], X_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
    }
}


__global__ void contract_tensor(
        Complex* state,
        const Complex* gate,
        int qubit,
        int* new_idx,
        int* old_idx,
        const int n,
        // const long long int N,
        const long long int lower,
        const long long int upper
) {
    extern __shared__ Complex shared_mem[]; // Use shared memory
    int idx = blockDim.x * blockIdx.x + threadIdx.x;


    // int chunk_size = upper-lower;
    int chunk_size = pow(2, n);

    if ((idx >= lower) & (idx < upper)) {

        int offset = idx * n;
        int temp = idx % chunk_size;

        // Compute the multi-dimensional index
        for (int i = n - 1; i >= 0; --i) {
            new_idx[offset + i] = temp % 2;
            temp /= 2;
        }

        // Copy new_idx to old_idx
        for (int i = 0; i < n; ++i) {
            old_idx[offset + i] = new_idx[offset + i];
        }

        // Compute the two values for j = 0 and j = 1 and store in shared memory
        for (int j = 0; j < 2; ++j) {
            old_idx[offset + qubit] = j;

            // Compute the linear index for old_idx
            int old_linear_idx = 0;
            int factor = 1;
            for (int i = n - 1; i >= 0; --i) {
                old_linear_idx += old_idx[offset + i] * factor;
                factor *= 2;
            }

            // needed to translate back to the full state array!!!
            if (idx >= chunk_size) {
                old_linear_idx += (idx / chunk_size) * chunk_size;
            }

            // Store the result in shared memory
            shared_mem[2*(idx % chunk_size) + j] = hipCmul(gate[new_idx[offset + qubit] * 2 + j], state[old_linear_idx]);
        }
        __syncthreads();
        state[idx] = hipCadd(shared_mem[2*(idx % chunk_size) + 1], shared_mem[2*(idx % chunk_size)]);

    }
}


__global__ void applyPhaseFlip(Complex* state, long long int idx) {
    state[idx] = hipCmul(state[idx], make_hipDoubleComplex(-1.0, 0.0));
}

void applyGateAllQubits(
    Complex* state,
    const Complex* gate,
    int* new_idx,
    int* old_idx,
    int n,
    dim3 dimBlock,
    dim3 dimGrid,
    int sharedMemSize,
    const long long int lower,
    const long long int upper
    ) {

    for (int i = 0; i < n; ++i) {
        contract_tensor<<<dimGrid, dimBlock, sharedMemSize>>>(state, gate, i, new_idx, old_idx, n, lower, upper);
    }
}

void applyGateSingleQubit(
    Complex* state,
    const Complex* gate,
    int* new_idx,
    int* old_idx,
    int n,
    long long int idx,
    dim3 dimBlock,
    dim3 dimGrid,
    int sharedMemSize,
    const long long int lower,
    const long long int upper
    ) {

    contract_tensor<<<dimGrid, dimBlock, sharedMemSize>>>(state, gate, idx, new_idx, old_idx, n, lower, upper);
}

void applyDiffusionOperator(
    Complex* state,
    const Complex* X_H,
    const Complex* H,
    const Complex* X,
    const Complex* Z,
    int* new_idx,
    int* old_idx,
    int n,
    dim3 dimBlock,
    dim3 dimGrid,
    int sharedMemSize,
    const long long int num_chunks_per_group,
    const long long int N_chunk,
    const long long int lower,
    const long long int upper
    ) {
    applyGateAllQubits(state, X_H, new_idx, old_idx, n, dimBlock, dimGrid, sharedMemSize, lower, upper);
    for (int i = 0; i < num_chunks_per_group; ++i) {
        applyPhaseFlip<<<dimGrid, dimBlock, 0>>>(state, (i+1)*N_chunk - 1);
    }

    applyGateSingleQubit(state, Z, new_idx, old_idx, n, 0, dimBlock, dimGrid, sharedMemSize, lower, upper);
    applyGateAllQubits(state, X, new_idx, old_idx, n, dimBlock, dimGrid, sharedMemSize, lower, upper);
    applyGateSingleQubit(state, Z, new_idx, old_idx, n, 0, dimBlock, dimGrid, sharedMemSize, lower, upper);
    applyGateAllQubits(state, H, new_idx, old_idx, n, dimBlock, dimGrid, sharedMemSize, lower, upper);
}
