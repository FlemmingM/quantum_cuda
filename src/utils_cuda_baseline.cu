
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <math.h>
#include <string.h>
#include <errno.h>
#include <omp.h>

typedef hipDoubleComplex Complex;

void saveArrayToCSV(const double *array, long long int N, const char* filename) {
    FILE *file = fopen(filename, "w");

    if (!file) {
        perror("Unable to open file");
        return;
    }
    fprintf(file, "position,probability\n");
    for (int i = 0; i < N; ++i) {
        fprintf(file, "pos%d,%f\n", i, array[i]);
    }
    fclose(file);
}


__device__ void AddComplex(hipDoubleComplex* a, hipDoubleComplex b){
  //transform the addresses of real and imag. parts to double pointers
  double *x = (double*)a;
  double *y = x+1;
  //use atomicAdd for double variables
  atomicAdd(x, hipCreal(b));
  atomicAdd(y, hipCimag(b));
}

__global__ void zeroOutState(Complex* new_state, long long int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) {
        new_state[idx] = make_hipDoubleComplex(0.0, 0.0);
    }
}


__global__ void updateState(Complex* state, Complex* new_state, long long int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) {
        state[idx] = new_state[idx];
    }
}

// BASELINE #########################
__global__ void contract_tensor_baseline(
        const Complex* state,
        const Complex* gate,
        int qubit,
        Complex* new_state,
        const int* shape,
        int* new_idx,
        int* old_idx,
        const int n,
        const long long int N
    ) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = idx * n;
    if (idx < N) {

        int temp = idx;

        // Compute the multi-dimensional index
        for (int i = n - 1; i >= 0; --i) {
            new_idx[offset+i] = temp % shape[i];
            temp /= shape[i];
        }

        // Perform the tensor contraction for the specified qubit
        for (int j = 0; j < 2; ++j) {
            // Copy new_idx to old_idx
            for (int i = 0; i < n; ++i) {
                old_idx[offset+i] = new_idx[offset+i];
            }
            old_idx[offset+qubit] = j;

            // Compute the linear index for old_idx
            int old_linear_idx = 0;
            int factor = 1;
            for (int i = n - 1; i >= 0; --i) {
                old_linear_idx += old_idx[offset+i] * factor;
                factor *= shape[i];
            }
            AddComplex(&new_state[idx], hipCmul(gate[new_idx[offset+qubit] * 2 + j], state[old_linear_idx]));
            }
        }
    }


void printState(const Complex* state, long long int N, const char* message) {
    printf("%s\n", message);
    for (int i = 0; i < N; ++i) {
        printf("(%.15f + %.15fi) ", hipCreal(state[i]), hipCimag(state[i]));
    }
    printf("\n");
}

__global__ void applyPhaseFlip(Complex* state, long long int idx) {
    state[idx] = hipCmul(state[idx], make_hipDoubleComplex(-1.0, 0.0));
}

void applyGateAllQubits(
    Complex* state,
    const Complex* gate,
    Complex* new_state,
    const int* shape,
    int* new_idx,
    int* old_idx,
    int n,
    long long int N,
    dim3 dimBlock,
    dim3 dimGrid
    ) {

    for (int i = 0; i < n; ++i) {
        contract_tensor_baseline<<<dimGrid, dimBlock>>>(state, gate, i, new_state, shape, new_idx, old_idx, n, N);
        updateState<<<dimGrid, dimBlock>>>(state, new_state, N);
        zeroOutState<<<dimGrid, dimBlock>>>(new_state, N);
    }
}

void applyGateSingleQubit(
    Complex* state,
    const Complex* gate,
    Complex* new_state,
    const int* shape,
    int* new_idx,
    int* old_idx,
    int n,
    long long int N,
    long long int idx,
    dim3 dimBlock,
    dim3 dimGrid
    ) {

    contract_tensor_baseline<<<dimGrid, dimBlock>>>(state, gate, idx, new_state, shape, new_idx, old_idx, n, N);
    // Update the state with the new state
    updateState<<<dimGrid, dimBlock>>>(state, new_state, N);
    zeroOutState<<<dimGrid, dimBlock>>>(new_state, N);
}

void applyDiffusionOperator(
    Complex* state,
    Complex* new_state,
    const int* shape,
    const Complex* H,
    const Complex* X,
    const Complex* Z,
    int* new_idx,
    int* old_idx,
    int n,
    long long int N,
    dim3 dimBlock,
    dim3 dimGrid
    ) {
    applyGateAllQubits(state, H, new_state, shape, new_idx, old_idx, n, N, dimBlock, dimGrid);
    applyGateAllQubits(state, X, new_state, shape, new_idx, old_idx, n, N, dimBlock, dimGrid);
    applyPhaseFlip<<<dimGrid, dimBlock>>>(state, N - 1);
    applyGateSingleQubit(state, Z, new_state, shape, new_idx, old_idx, n, N, 0, dimBlock, dimGrid);
    applyGateAllQubits(state, X, new_state, shape, new_idx, old_idx, n, N, dimBlock, dimGrid);
    applyGateSingleQubit(state, Z, new_state, shape, new_idx, old_idx, n, N, 0, dimBlock, dimGrid);
    applyGateAllQubits(state, H, new_state, shape, new_idx, old_idx, n, N, dimBlock, dimGrid);
}
