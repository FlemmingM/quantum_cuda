#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <math.h>
#include <string.h>
#include <errno.h>
#include <omp.h>
#include "utils_cuda_opt7.h"

typedef hipDoubleComplex Complex;

int main(int argc, char* argv[]) {

    // collect input args
    // if (argc < 6) {
    //     fprintf(stderr, "Usage: %s n qubits<int>; marked state<int>; number of samples<int>; fileName<string>; verbose 0 or 1<int>\n", argv[0]);
    //     return 1;
    // }

    int n = atoi(argv[1]);
    long long int N = (long long int)pow(2, n);
    long long int markedState = atoi(argv[2]);
    // int numSamples = atoi(argv[3]);
    // const char* fileName = argv[4];
    // int verbose = atoi(argv[5]);

    if (markedState > (N-1)) {
        fprintf(stderr, "You chose a markedState %d but the largest state possible is state %d", markedState, (N-1));
        return 1;
    }

    // Define the gates
    hipDoubleComplex H_h[4] = {
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0),
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(-1.0 / sqrt(2.0), 0.0)
    };

    hipDoubleComplex X_H_h[4] = {
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(-1.0 / sqrt(2.0), 0.0),
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0)
    };

    hipDoubleComplex I_h[4] = {
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0),
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(1.0, 0.0)
    };

    hipDoubleComplex Z_h[4] = {
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0),
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(-1.0, 0.0)
    };

    hipDoubleComplex X_h[4] = {
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(1.0, 0.0),
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0)
    };

    Complex *state_h;
    Complex *state_d;
    // Complex *new_state_h;
    // Complex *new_state_d;
    Complex *H_d;
    Complex *X_H_d;
    Complex *I_d;
    Complex *Z_d;
    Complex *X_d;

    int *new_idx_d;
    int *old_idx_d;


    // Init the state
    hipHostMalloc((void **)&state_h, N * sizeof(Complex));
    hipMalloc((void **)&state_d, N * sizeof(Complex));
    // Init the |0>^(xn) state and the new_state
    // state_h[0] = make_hipDoubleComplex(1.0, 0.0);
    // for (int i = 1; i < N; ++i) {
    //     state_h[i] = make_hipDoubleComplex(0.0, 0.0);
    // }
    // hipMemcpy(state_d, state_h, N * sizeof(Complex), hipMemcpyHostToDevice);


    // Malloc the gate on device
    hipMalloc((void **)&H_d, 4 * sizeof(Complex));
    hipMalloc((void **)&X_H_d, 4 * sizeof(Complex));
    hipMalloc((void **)&I_d, 4 * sizeof(Complex));
    hipMalloc((void **)&Z_d, 4 * sizeof(Complex));
    hipMalloc((void **)&X_d, 4 * sizeof(Complex));



    // Copy from host to device

    hipMemcpy(H_d, H_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
    hipMemcpy(X_H_d, X_H_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
    hipMemcpy(I_d, I_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
    hipMemcpy(Z_d, Z_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
    hipMemcpy(X_d, X_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);

    int val = 256;
    dim3 dimBlock(val);
    dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x);

    const int blockSize = val;
    const int gridSize = (N + blockSize - 1) / blockSize;

    // Allocate shared memory for reduction
    // int sharedMemSize = blockSize * sizeof(Complex);
    int sharedMemSize = 2*N * sizeof(Complex);

    // Malloc the indices on the device
    hipMalloc(&new_idx_d, gridSize * blockSize * n * sizeof(int));
    hipMalloc(&old_idx_d, gridSize * blockSize * n * sizeof(int));


    // Assuming we have t = 1 solution in grover's algorithm
    // we have k = floor(pi/4 * sqrt(N))
    int k = (int)floor(M_PI / 4 * sqrt(N));


    // https://forums.developer.nvidia.com/t/question-about-max-shared-memory-in-block-and-multiprocessor/283345

    hipFuncSetAttribute(reinterpret_cast<const void*>(contract_tensor), hipFuncAttributeMaxDynamicSharedMemorySize, sharedMemSize);
    double time = omp_get_wtime();
    zeroOutState<<<gridSize, blockSize>>>(state_d, N);

    // contract_tensor<<<gridSize, blockSize, sharedMemSize>>>(state_d, H_d, 0, shape_d, new_idx_d, old_idx_d, n, N);
    // contract_tensor<<<gridSize, blockSize>>>(state_d, H_d, 0, new_state_d, shape_d, new_idx_d, old_idx_d, n, N);

        // contract_tensor_baseline<<<dimGrid, dimBlock>>>(state, gate, i, new_state, shape, n, N);
        // hipDeviceSynchronize();
        // Update the state with the new state
    // updateState<<<gridSize, blockSize>>>(state_d, new_state_d, N);





    // Now apply the H gate n times, once for each qubit
    applyGateAllQubits(state_d, H_d, new_idx_d, old_idx_d, n, N, dimBlock, dimGrid, sharedMemSize);

    // hipDeviceSynchronize();


    // Apply Grover's algorithm k iteration and then sample
    // if (verbose == 1) {
    //     printf("Running %d round(s)\n", k);
    // }

    // for (int i = 0; i < k; ++i) {
    //     applyPhaseFlip<<<dimGrid, dimBlock>>>(state_d, markedState);
    //     applyDiffusionOperator(state_d, X_H_d, H_d, X_d, Z_d, new_idx_d, old_idx_d, n, N, dimBlock, dimGrid, sharedMemSize);
    //     // hipDeviceSynchronize();
    // }

    hipDeviceSynchronize();
    double elapsed = omp_get_wtime() - time;
    printf("Time: %f \n", elapsed);


    hipMemcpy(state_h, state_d, N * sizeof(Complex), hipMemcpyDeviceToHost);

    // if (verbose == 1) {
    printState(state_h, N, "Initial state");
    // }

    // // Apply Grover's algorithm k iteration and then sample
    // if (verbose == 1) {
    //     printf("Running %d round(s)\n", k);
    // }

    // double time = omp_get_wtime();

    // for (int i = 0; i < k; ++i) {
    //     if (verbose == 1) {
    //         printf("%d/%d\n", i, k);
    //     }
    //     // Apply Oracle
    //     applyPhaseFlip(state, markedState);
    //     if (verbose == 1) {
    //         printState(state, N, "Oracle applied");
    //     }
    //     // Apply the diffusion operator
    //     applyDiffusionOperator(state, new_state, shape, H, X, Z, n, N);
    //     if (verbose == 1) {
    //         printState(state, N, "After Diffusion");
    //     }
    // }

    // double elapsed = omp_get_wtime() - time;
    // printf("Time: %f \n", elapsed);

    // // Sample the states wheighted by their amplitudes
    // double* averages = simulate(state_h, N, 1);
    // if (verbose == 1) {
    //     printf("Average frequency per position:\n");
    //     for (int i = 0; i < N; ++i) {
    //         printf("Position %d: %f\n", i, averages[i]);
    //     }
    // }


    // // save the data
    // saveArrayToCSV(averages, N, fileName);

    hipFree(state_d);
    hipFree(H_d);

    hipHostFree(state_h);
    hipHostFree(H_h);
    hipHostFree(I_h);
    hipHostFree(Z_h);
    hipHostFree(X_h);

    return 0;
}
