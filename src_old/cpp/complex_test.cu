
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
// #include <cuda_runtime.h>

typedef hipDoubleComplex Complex;


__global__
void addNums(const Complex *a, const Complex *b, Complex *c, int N){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < N) {
        c[idx] = hipCadd(a[idx], b[idx]);
    }
}

int main() {
    int N = 10;
    size_t size = N * sizeof(hipDoubleComplex);

    // Host memory allocation with cudaMallocHost
    hipDoubleComplex *h_a, *h_b, *h_c;
    hipHostMalloc((void **)&h_a, size, hipHostMallocDefault);
    hipHostMalloc((void **)&h_b, size, hipHostMallocDefault);
    hipHostMalloc((void **)&h_c, size, hipHostMallocDefault);

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        h_a[i] = make_hipDoubleComplex(i, i);
        h_b[i] = make_hipDoubleComplex(i, -i);
    }

    hipDoubleComplex *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy data from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    addNums<<<numBlocks, blockSize>>>(d_a, d_b, d_c, N);

    // Copy results from device to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Print the results
    printf("Results of complex addition:\n");
    for (int i = 0; i < N; i++) {
        printf("(%f, %f) + (%f, %f) = (%f, %f)\n",
               hipCreal(h_a[i]), hipCimag(h_a[i]),
               hipCreal(h_b[i]), hipCimag(h_b[i]),
               hipCreal(h_c[i]), hipCimag(h_c[i]));
    }

    // Free memory
    hipFree(h_a);
    hipFree(h_b);
    hipFree(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);


}



