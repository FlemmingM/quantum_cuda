#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <math.h>
#include <string.h>
#include <errno.h>
#include <omp.h>
#include "utils_cuda_opt2.h"

typedef hipDoubleComplex Complex;

int main(int argc, char* argv[]) {

    // collect input args
    // if (argc < 6) {
    //     fprintf(stderr, "Usage: %s n qubits<int>; marked state<int>; number of samples<int>; fileName<string>; verbose 0 or 1<int>\n", argv[0]);
    //     return 1;
    // }

    int n = atoi(argv[1]);
    long long int N = (long long int)pow(2, n);
    long long int markedState = atoi(argv[2]);
    // int numSamples = atoi(argv[3]);
    // const char* fileName = argv[4];
    // int verbose = atoi(argv[5]);

    if (markedState > (N-1)) {
        fprintf(stderr, "You chose a markedState %d but the largest state possible is state %d", markedState, (N-1));
        return 1;
    }

    // Define the gates
    hipDoubleComplex H_h[4] = {
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0),
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(-1.0 / sqrt(2.0), 0.0)
    };

    hipDoubleComplex I_h[4] = {
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0),
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(1.0, 0.0)
    };

    hipDoubleComplex Z_h[4] = {
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0),
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(-1.0, 0.0)
    };

    hipDoubleComplex X_h[4] = {
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(1.0, 0.0),
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0)
    };

    Complex *state_h;
    Complex *state_d;
    Complex *new_state_h;
    Complex *new_state_d;
    Complex *H_d;
    Complex *I_d;
    Complex *Z_d;
    Complex *X_d;

    int *shape_h;
    int *shape_d;
    int *new_idx_d;
    int *old_idx_d;

    // Malloc on device and host

    // Init the temp new state for the results
    hipHostMalloc((void **)&new_state_h, N * sizeof(Complex));
    hipMalloc((void **)&new_state_d, N * sizeof(Complex));
    for (int i = 0; i < N; ++i) {
        new_state_h[i] = make_hipDoubleComplex(0.0, 0.0);
    }
    hipMemcpy(new_state_d, new_state_h, N * sizeof(Complex), hipMemcpyHostToDevice);

    // We don't need it in on the host
    hipHostFree(new_state_h);



    // Init the state
    hipHostMalloc((void **)&state_h, N * sizeof(Complex));
    hipMalloc((void **)&state_d, N * sizeof(Complex));
    // Init the |0>^(xn) state and the new_state
    state_h[0] = make_hipDoubleComplex(1.0, 0.0);
    for (int i = 1; i < N; ++i) {
        state_h[i] = make_hipDoubleComplex(0.0, 0.0);
    }
    hipMemcpy(state_d, state_h, N * sizeof(Complex), hipMemcpyHostToDevice);

    hipHostMalloc((void **)&shape_h, n * sizeof(int));
    hipMalloc((void **)&shape_d, n * sizeof(int));

    // Malloc the gate on device
    hipMalloc((void **)&H_d, 4 * sizeof(Complex));
    hipMalloc((void **)&I_d, 4 * sizeof(Complex));
    hipMalloc((void **)&Z_d, 4 * sizeof(Complex));
    hipMalloc((void **)&X_d, 4 * sizeof(Complex));




    // Init the shape depending on the number of qubits
    // each qubit is a column vector of size 2
    // e.g. |0> = [1, 0]
    // Thus, for n=3 qubits (N=8) the tensor will have a shape of 2,2,2
    for (int i = 0; i < n; ++i) {
        shape_h[i] = 2;
    }




    // Copy from host to device
    hipMemcpy(shape_d, shape_h, n * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(H_d, H_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
    hipMemcpy(I_d, I_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
    hipMemcpy(Z_d, Z_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
    hipMemcpy(X_d, X_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);


    dim3 dimBlock(256);
    dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x);

    const int blockSize = 256;
    const int gridSize = (N + blockSize - 1) / blockSize;

    // Allocate shared memory for reduction
    // int sharedMemSize = blockSize * sizeof(Complex);
    int sharedMemSize = 2*N * sizeof(Complex);

    // Malloc the indices on the device
    hipMalloc(&new_idx_d, gridSize * blockSize * n * sizeof(int));
    hipMalloc(&old_idx_d, gridSize * blockSize * n * sizeof(int));


    // Assuming we have t = 1 solution in grover's algorithm
    // we have k = floor(pi/4 * sqrt(N))
    int k = (int)floor(M_PI / 4 * sqrt(N));



    double time = omp_get_wtime();


    // contract_tensor<<<gridSize, blockSize, sharedMemSize>>>(state_d, H_d, 0, new_state_d, shape_d, new_idx_d, old_idx_d, n, N);
    // contract_tensor<<<gridSize, blockSize>>>(state_d, H_d, 0, new_state_d, shape_d, new_idx_d, old_idx_d, n, N);

        // contract_tensor_baseline<<<dimGrid, dimBlock>>>(state, gate, i, new_state, shape, n, N);
        // hipDeviceSynchronize();
        // Update the state with the new state
    // updateState<<<gridSize, blockSize>>>(state_d, new_state_d, N);





    // Now apply the H gate n times, once for each qubit
    applyGateAllQubits(state_d, H_d, new_state_d, shape_d, new_idx_d, old_idx_d, n, N, dimBlock, dimGrid, sharedMemSize);

    // hipDeviceSynchronize();


    // Apply Grover's algorithm k iteration and then sample
    // if (verbose == 1) {
    //     printf("Running %d round(s)\n", k);
    // }

    for (int i = 0; i < k; ++i) {
        applyPhaseFlip<<<dimGrid, dimBlock>>>(state_d, markedState);
        applyDiffusionOperator(state_d, new_state_d, shape_d, H_d, X_d, Z_d, new_idx_d, old_idx_d, n, N, dimBlock, dimGrid, sharedMemSize);
        // hipDeviceSynchronize();
    }

    hipDeviceSynchronize();
    double elapsed = omp_get_wtime() - time;
    printf("Time: %f \n", elapsed);


    hipMemcpy(state_h, state_d, N * sizeof(Complex), hipMemcpyDeviceToHost);

    // if (verbose == 1) {
    printState(state_h, N, "Initial state");
    // }

    // // Apply Grover's algorithm k iteration and then sample
    // if (verbose == 1) {
    //     printf("Running %d round(s)\n", k);
    // }

    // double time = omp_get_wtime();

    // for (int i = 0; i < k; ++i) {
    //     if (verbose == 1) {
    //         printf("%d/%d\n", i, k);
    //     }
    //     // Apply Oracle
    //     applyPhaseFlip(state, markedState);
    //     if (verbose == 1) {
    //         printState(state, N, "Oracle applied");
    //     }
    //     // Apply the diffusion operator
    //     applyDiffusionOperator(state, new_state, shape, H, X, Z, n, N);
    //     if (verbose == 1) {
    //         printState(state, N, "After Diffusion");
    //     }
    // }

    // double elapsed = omp_get_wtime() - time;
    // printf("Time: %f \n", elapsed);

    // // Sample the states wheighted by their amplitudes
    // double* averages = simulate(state, N, numSamples);
    // if (verbose == 1) {
    //     printf("Average frequency per position:\n");
    //     for (int i = 0; i < N; ++i) {
    //         printf("Position %d: %f\n", i, averages[i]);
    //     }
    // }


    // // save the data
    // saveArrayToCSV(averages, N, fileName);

    hipFree(state_d);
    hipFree(new_state_d);
    hipFree(shape_d);
    hipFree(H_d);
    hipHostFree(state_h);

    hipHostFree(shape_h);

    hipHostFree(H_h);
    hipHostFree(I_h);
    hipHostFree(Z_h);
    hipHostFree(X_h);

    return 0;
}
