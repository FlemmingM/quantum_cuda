#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <math.h>
#include <string.h>
#include <errno.h>
#include <omp.h>
#include "utils_cuda_opt8.h"

typedef hipDoubleComplex Complex;

int main(int argc, char* argv[]) {

    // collect input args
    // if (argc < 6) {
    //     fprintf(stderr, "Usage: %s n qubits<int>; marked state<int>; number of samples<int>; fileName<string>; verbose 0 or 1<int>\n", argv[0]);
    //     return 1;
    // }

    int n = atoi(argv[1]);
    long long int N = (long long int)pow(2, n);
    long long int markedState = atoi(argv[2]);
    int warp_size = atoi(argv[3]);
    // int numSamples = atoi(argv[3]);
    // const char* fileName = argv[4];
    // int verbose = atoi(argv[5]);

    if (markedState > (N-1)) {
        fprintf(stderr, "You chose a markedState %d but the largest state possible is state %d", markedState, (N-1));
        return 1;
    }

    // Define the gates
    hipDoubleComplex H_h[4] = {
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0),
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(-1.0 / sqrt(2.0), 0.0)
    };

    hipDoubleComplex X_H_h[4] = {
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(-1.0 / sqrt(2.0), 0.0),
        make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0), make_hipDoubleComplex(1.0 / sqrt(2.0), 0.0)
    };

    hipDoubleComplex I_h[4] = {
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0),
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(1.0, 0.0)
    };

    hipDoubleComplex Z_h[4] = {
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0),
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(-1.0, 0.0)
    };

    hipDoubleComplex X_h[4] = {
        make_hipDoubleComplex(0.0, 0.0), make_hipDoubleComplex(1.0, 0.0),
        make_hipDoubleComplex(1.0, 0.0), make_hipDoubleComplex(0.0, 0.0)
    };

    Complex *state_h;
    Complex *state_d;
    Complex *H_d;
    Complex *X_H_d;
    Complex *I_d;
    Complex *Z_d;
    Complex *X_d;

    int *new_idx_d;
    int *old_idx_d;
    int *old_linear_idxs_h;
    int *old_linear_idxs_d;
    int *shared_idxs_d;
    int *shared_idxs_h;

    // Init the state
    hipHostMalloc((void **)&state_h, N * sizeof(Complex));
    hipMalloc((void **)&state_d, N * sizeof(Complex));
    // Init the |0>^(xn) state and the new_state
    state_h[0] = make_hipDoubleComplex(1.0, 0.0);
    for (int i = 1; i < N; ++i) {
        state_h[i] = make_hipDoubleComplex(0.0, 0.0);
    }
    hipMemcpy(state_d, state_h, N * sizeof(Complex), hipMemcpyHostToDevice);


    // Malloc the gate on device
    hipMalloc((void **)&H_d, 4 * sizeof(Complex));
    hipMalloc((void **)&X_H_d, 4 * sizeof(Complex));
    hipMalloc((void **)&I_d, 4 * sizeof(Complex));
    hipMalloc((void **)&Z_d, 4 * sizeof(Complex));
    hipMalloc((void **)&X_d, 4 * sizeof(Complex));

    // Copy from host to device
    hipMemcpy(H_d, H_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
    hipMemcpy(X_H_d, X_H_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
    hipMemcpy(I_d, I_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
    hipMemcpy(Z_d, Z_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);
    hipMemcpy(X_d, X_h, 4 * sizeof(Complex), hipMemcpyHostToDevice);


    dim3 dimBlock(256);
    dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x);

    const int blockSize = 256;
    const int gridSize = (N + blockSize - 1) / blockSize;

    // Allocate shared memory for reduction
    // int sharedMemSize = blockSize * sizeof(Complex);
    int sharedMemSize = 2*N * sizeof(Complex);
    // int sharedMemSize2 = 32 * 2 * N * sizeof(int);

        // Check if the requested shared memory size exceeds the limit
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    if (sharedMemSize > deviceProp.sharedMemPerBlock) {
        printf("Requested shared memory size (%d bytes) exceeds the device limit (%d bytes).\n", sharedMemSize, deviceProp.sharedMemPerBlock);
        return -1;
    }

    // Malloc the indices on the device
    hipMalloc(&new_idx_d, gridSize * blockSize * n * sizeof(int));
    hipMalloc(&old_idx_d, gridSize * blockSize * n * sizeof(int));

    hipHostMalloc(&old_linear_idxs_h, 2 * N * n * sizeof(int));
    hipMalloc(&old_linear_idxs_d, 2 * N * n * sizeof(int));

    hipMalloc(&shared_idxs_d, warp_size * 2 * N * sizeof(int));
    hipHostMalloc(&shared_idxs_h, warp_size * 2 * N * sizeof(int));

    // Assuming we have t = 1 solution in grover's algorithm
    // we have k = floor(pi/4 * sqrt(N))
    int k = (int)floor(M_PI / 4 * sqrt(N));

    zeroOutState<<<gridSize, blockSize>>>(shared_idxs_h, warp_size * 2 * N);


    hipMemcpy(shared_idxs_d, shared_idxs_h, warp_size*2*N*sizeof(int), hipMemcpyHostToDevice);
    double time = omp_get_wtime();

    for (int i = 0; i < n; ++i) {
        zeroOutState<<<gridSize, blockSize>>>(shared_idxs_d, warp_size * 2 * N);
        compute_idx<<<gridSize, blockSize>>>(i, new_idx_d, old_idx_d, n, N, shared_idxs_d, warp_size);
        warp_sum_reduction<<<(N*2*warp_size + blockSize - 1) / blockSize, blockSize>>>(shared_idxs_d, old_linear_idxs_d + 2*i*N, warp_size * 2 * N, warp_size);
    }


    hipDeviceSynchronize();
    // hipMemcpy(shared_idxs_h, shared_idxs_d, warp_size*2*N*sizeof(int), hipMemcpyDeviceToHost);
    // hipMemcpy(old_linear_idxs_h, old_linear_idxs_d, 2*N* n * sizeof(int), hipMemcpyDeviceToHost);


    // for (int i = 0; i < (2*N*n); ++i) {
    //     printf("%d ", old_linear_idxs_h[i]);
    // }

    // printf("###\n");
    // for (int i = 0; i < (warp_size*2*N); ++i) {
    //     printf("%d ", shared_idxs_h[i]);
    // }
    // contract_tensor<<<(N*2 + blockSize - 1) / blockSize, blockSize, sharedMemSize>>>(state_d, H_d, 0, new_idx_d, old_idx_d, n, N, old_linear_idxs_d);
    // contract_tensor<<<(N*2 + blockSize - 1) / blockSize, blockSize, sharedMemSize>>>(state_d, H_d, 1, new_idx_d, old_idx_d, n, N, old_linear_idxs_d);
    // contract_tensor<<<(N*2 + blockSize - 1) / blockSize, blockSize, sharedMemSize>>>(state_d, H_d, 2, new_idx_d, old_idx_d, n, N, old_linear_idxs_d);

    // contract_tensor<<<gridSize, blockSize, sharedMemSize>>>(state_d, H_d, 1, new_idx_d, old_idx_d, n, N, old_linear_idxs_d);
    // contract_tensor<<<gridSize, blockSize, sharedMemSize>>>(state_d, H_d, 2, new_idx_d, old_idx_d, n, N, old_linear_idxs_d);

    // Now apply the H gate n times, once for each qubit
    // applyGateAllQubits(state_d, H_d, new_idx_d, old_idx_d, n, N, dimBlock, (N*2 + blockSize - 1) / blockSize, sharedMemSize, old_linear_idxs_d);
    // for (int i = 0; i < k; ++i) {
    //     applyPhaseFlip<<<dimGrid, dimBlock>>>(state_d, markedState);
    //     applyDiffusionOperator(state_d, X_H_d, H_d, X_d, Z_d, new_idx_d, old_idx_d, n, N, dimBlock, (N*2 + blockSize - 1) / blockSize, sharedMemSize, old_linear_idxs_d);
    // }

    applyGateAllQubits(state_d, H_d, new_idx_d, old_idx_d, n, N, dimBlock, dimGrid, sharedMemSize, old_linear_idxs_d);
    // Apply Grover's algorithm k iteration and then sample
    // for (int i = 0; i < k; ++i) {
    //     applyPhaseFlip<<<dimGrid, dimBlock>>>(state_d, markedState);
    //     applyDiffusionOperator(state_d, X_H_d, H_d, X_d, Z_d, new_idx_d, old_idx_d, n, N, dimBlock, dimGrid, sharedMemSize, old_linear_idxs_d);
    // }

    hipDeviceSynchronize();
    double elapsed = omp_get_wtime() - time;
    printf("Time: %f \n", elapsed);


    hipMemcpy(state_h, state_d, N * sizeof(Complex), hipMemcpyDeviceToHost);

    printState(state_h, N, "Initial state");



    hipFree(state_d);
    hipFree(H_d);
    hipFree(I_d);
    hipFree(Z_d);
    hipFree(X_d);
    hipFree(X_H_d);
    hipFree(shared_idxs_d);
    hipFree(old_linear_idxs_d);

    hipHostFree(state_h);
    hipHostFree(shared_idxs_h);
    hipHostFree(old_linear_idxs_h);

    return 0;
}
