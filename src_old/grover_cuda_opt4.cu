#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <math.h>
#include <string.h>
#include <errno.h>
#include <omp.h>
#include "utils_cuda_opt4.h"

typedef hipDoubleComplex Complex;


#define cudaCheckError(call) {                               \
    hipError_t err = call;                                  \
    if (err != hipSuccess) {                                \
        fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n", \
                __FILE__, __LINE__, hipGetErrorString(err));\
        exit(EXIT_FAILURE);                                  \
    }                                                        \
}



int main(int argc, char* argv[]) {

    // collect input args
    // if (argc < 6) {
    //     fprintf(stderr, "Usage: %s n qubits<int>; marked state<int>; number of samples<int>; fileName<string>; verbose 0 or 1<int>\n", argv[0]);
    //     return 1;
    // }

    int n = atoi(argv[1]);
    long long int N = (long long int)pow(2, n);
    long long int markedState = atoi(argv[2]);
    const int num_chunks_per_group = atoi(argv[3]);
    const int num_qubits_per_group = atoi(argv[4]);
    // const int block_size = atoi(argv[4]);
    // const char* fileName = argv[4];
    // int verbose = atoi(argv[5]);

    if (markedState > (N-1)) {
        fprintf(stderr, "You chose a markedState %d but the largest state possible is state %d", markedState, (N-1));
        return 1;
    }


    // Define the number of groups to do the parallel search with more than 10 qubits
    // while still using the fast shared memory

    long long int num_groups = N / pow(2, num_qubits_per_group);
    int num_qubits_per_chunk = num_qubits_per_group - (int)log2(num_chunks_per_group);
    int N_chunk = pow(2, num_qubits_per_chunk);
    long long int num_chunks = num_groups * num_chunks_per_group;
    printf("N: %lld\n", N);
    printf("n: %d\n", n);
    printf("num_groups: %lld\n", num_groups);
    printf("num_chunks_per_group: %d\n", num_chunks_per_group);
    printf("num_qubits_per_chunk: %d\n", num_qubits_per_chunk);
    printf("N_chunk: %d\n", N_chunk);
    printf("num_chunks: %lld\n", num_chunks);

    if (N_chunk > pow(2, 10)) {
        fprintf(stderr, "You chose a number of qubits per group of: %d and a number of chunks per group of: %d\n Change the config so that the number of qubits per chunk is maximally 10 to fit into 1 block", num_qubits_per_group, num_chunks_per_group);
        return 1;
    }

    int sharedMemSize = (int)(pow(2, 11)) * sizeof(Complex);


    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    if (sharedMemSize > deviceProp.sharedMemPerBlock) {
        printf("Requested shared memory size (%d bytes) exceeds the device limit (%d bytes).\n", sharedMemSize, deviceProp.sharedMemPerBlock);
        return -1;
    }


    long long int oracle_chunk = markedState / (N / num_chunks);


    markedState = markedState % (N / num_chunks);
    long long int recoveredState = oracle_chunk*(N / num_chunks)+markedState;
    printf("oracle_chunk: %lld, pos: %lld, recovered: %lld\n", oracle_chunk, markedState, recoveredState);


    dim3 dimBlock(N_chunk);
    dim3 dimGrid(num_chunks_per_group);

    printf("dimGrid: %d, dimBlock: %d\n", dimGrid.x, dimBlock.x);

    // Set the gates:
    int num_devices = 1;
    Complex *H_d[num_devices];
    Complex *I_d[num_devices];
    Complex *Z_d[num_devices];
    Complex *X_d[num_devices];
    Complex *X_H_d[num_devices];
    allocateGatesDevice(num_devices, H_d, I_d, Z_d, X_d, X_H_d);



    // // Assuming we have t = 1 solution in grover's algorithm
    // // we have k = floor(pi/4 * sqrt(N/num_chunks))
    long long int k = (int)floor(M_PI / 4 * sqrt(N/num_chunks));
    printf("running %lld rounds\n", k);



    double time = omp_get_wtime();

    hipStream_t streams[num_chunks_per_group];

    Complex *solution_state_h;
    Complex *state_h[num_chunks_per_group];
    Complex *state_d[num_chunks_per_group];
    int *new_idx_d[num_chunks_per_group];
    int *old_idx_d[num_chunks_per_group];

    // To get the parallel search results
    int *d_maxIndex;
    int *h_maxIndex;
    int *d_chunk_ids;
    int *h_chunk_ids;
    double *d_maxValue;
    double *h_maxValue;
    hipMalloc((void**)&d_maxIndex, num_chunks_per_group*sizeof(int));
    hipHostMalloc((void**)&h_maxIndex, num_chunks_per_group*sizeof(int));
    hipMalloc((void**)&d_maxValue, num_chunks_per_group*sizeof(double));
    hipHostMalloc((void**)&h_maxValue, num_chunks_per_group*sizeof(double));
    hipMalloc((void**)&d_chunk_ids, num_chunks_per_group*sizeof(double));
    hipHostMalloc((void**)&h_chunk_ids, num_chunks_per_group*sizeof(double));


    // Create the streams
    for (int i = 0; i < num_chunks_per_group; ++i) {
        hipStreamCreate(&streams[i]);
        cudaCheckError(hipHostMalloc((void **)&state_h[i], N_chunk * sizeof(Complex)));
        state_h[i][0] = make_hipDoubleComplex(1.0, 0.0);
        for (int idx = 1; idx < N_chunk; ++idx) {
            state_h[i][idx] = make_hipDoubleComplex(0.0, 0.0);
        }

        hipMalloc(&new_idx_d[i], N_chunk * num_qubits_per_chunk * sizeof(int));
        hipMalloc(&old_idx_d[i], N_chunk * num_qubits_per_chunk * sizeof(int));
        hipMalloc((void **)&state_d[i], N_chunk * sizeof(Complex));
        cudaCheckError(hipMemcpyAsync(state_d[i], state_h[i], N_chunk * sizeof(Complex), hipMemcpyHostToDevice, streams[i]));
        hipMemcpyAsync(h_maxIndex, d_maxIndex, sizeof(int), hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(h_chunk_ids, d_chunk_ids, sizeof(int), hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(h_maxValue, d_maxValue, sizeof(double), hipMemcpyHostToDevice, streams[i]);
    }


    // allocate the solution state:
    hipHostMalloc((void **)&solution_state_h, N_chunk * sizeof(Complex));

    int marked_chunk = -99;
    for (int j = 0; j < num_groups; ++j) {
        // printf("%d / %d\n", j, num_groups);
        // #pragma omp parallel for num_threads(num_groups)
        for (int i = 0; i < num_chunks_per_group; ++i) {
            // hipStreamCreate(&streams[i]);
            int index = j*num_chunks_per_group+i;

            // ### Here we run Grover's algorithm
            // initState<<<dimGrid, dimBlock, 0, streams[i]>>>(state_d[i], N_chunk);
            applyGateAllQubits(
                state_d[i],
                H_d[0], new_idx_d[i],
                old_idx_d[i], num_qubits_per_chunk,
                dimBlock,
                dimGrid,
                sharedMemSize,
                0,
                N_chunk,
                streams[i]
            );
            for (int l = 0; l < k; ++l) {
                if (oracle_chunk == (index)) {
                    // printf("oracle chunk_id: %d\n", index);
                    applyPhaseFlip<<<dimGrid, dimBlock, 0, streams[i]>>>(state_d[i], markedState);
                }

                applyDiffusionOperator(
                    state_d[i],
                    X_H_d[0], H_d[0], X_d[0], Z_d[0], new_idx_d[i],
                    old_idx_d[i], num_qubits_per_chunk, dimBlock, dimGrid, sharedMemSize,
                    0, N_chunk,
                    streams[i]
                );
            }

            hipStreamSynchronize(streams[i]);

        }
        hipDeviceSynchronize();


        for (int i = 0; i < num_chunks_per_group; ++i){
            int index = j*num_chunks_per_group+i;
            findMaxIndexKernel<<<1, N_chunk, 0, streams[i]>>>(state_d[i], d_maxIndex, d_maxValue, N_chunk, index, d_chunk_ids);
            hipMemcpyAsync(h_maxIndex, d_maxIndex, sizeof(int), hipMemcpyDeviceToHost, streams[i]);
            hipMemcpyAsync(h_chunk_ids, d_chunk_ids, sizeof(int), hipMemcpyDeviceToHost, streams[i]);
            hipMemcpyAsync(h_maxValue, d_maxValue, sizeof(double), hipMemcpyDeviceToHost, streams[i]);
            // hipMemcpyAsync(state_h[i], state_d[i], N_chunk * sizeof(Complex), hipMemcpyDeviceToHost, streams[i]);
            hipStreamSynchronize(streams[i]);
        }
        hipDeviceSynchronize();


        for (int i = 0; i < num_chunks_per_group; ++i){
            if(h_maxValue[i] >= 0.7){
                printf("chunk id: %d, maxIdx: %d, maxVal: %f\n", h_chunk_ids[i], h_maxIndex[i], h_maxValue[i]);
                marked_chunk = h_chunk_ids[i];
                int index = marked_chunk % num_chunks_per_group;

                hipMemcpyAsync(solution_state_h, state_d[index], N_chunk * sizeof(Complex), hipMemcpyDeviceToHost, streams[index]);
                hipStreamSynchronize(streams[index]);
            }
        }
        hipDeviceSynchronize();

        for (int i = 0; i < num_chunks_per_group; ++i){
            initState<<<dimGrid, dimBlock, 0, streams[i]>>>(state_d[i], N_chunk);
            cudaCheckError(hipStreamSynchronize(streams[i]));
        }
        hipDeviceSynchronize();

    } // end of the out loop

    for (int i = 0; i < num_chunks_per_group; ++i) {
        hipStreamDestroy(streams[i]);
    }

    double elapsed = omp_get_wtime() - time;
    printf("Time: %f \n", elapsed);

    // printState(solution_state_h, N_chunk, "Initial state");


    for (int i = 0; i < num_devices; ++i) {
        hipFree(H_d[i]);
        hipFree(I_d[i]);
        hipFree(Z_d[i]);
        hipFree(X_d[i]);
        hipFree(X_H_d[i]);
    }


    for (int i = 0; i < num_chunks_per_group; ++i) {
        hipFree(new_idx_d[i]);
        hipFree(old_idx_d[i]);
        hipFree(state_d[i]);
        hipHostFree(state_h[i]);


    }

    hipHostFree(solution_state_h);
    hipFree(d_maxIndex);
    hipFree(d_chunk_ids);
    hipFree(d_maxValue);
    hipHostFree(h_maxIndex);
    hipHostFree(h_chunk_ids);
    hipHostFree(h_maxValue);

    return 0;
}
