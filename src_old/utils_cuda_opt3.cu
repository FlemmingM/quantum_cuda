
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <math.h>
#include <string.h>
#include <errno.h>
#include <omp.h>

typedef hipDoubleComplex Complex;

void printState(const Complex* state, long long int N, const char* message) {
    printf("%s\n", message);
    for (int i = 0; i < N; ++i) {
        printf("(%.15f + %.15fi) ", hipCreal(state[i]), hipCimag(state[i]));
    }
    printf("\n");
}

void saveArrayToCSV(const double *array, long long int N, const char* filename) {
    FILE *file = fopen(filename, "w");

    if (!file) {
        perror("Unable to open file");
        return;
    }
    fprintf(file, "position,probability\n");
    for (int i = 0; i < N; ++i) {
        fprintf(file, "pos%d,%f\n", i, array[i]);
    }
    fclose(file);
}


__device__ void AddComplex(hipDoubleComplex* a, hipDoubleComplex b){
  //transform the addresses of real and imag. parts to double pointers
  double *x = (double*)a;
  double *y = x+1;
  //use atomicAdd for double variables
  atomicAdd(x, hipCreal(b));
  atomicAdd(y, hipCimag(b));
}

__global__ void zeroOutState(Complex* new_state, long long int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) {
        new_state[idx] = make_hipDoubleComplex(0.0, 0.0);
    }
}


__global__ void updateState(Complex* state, Complex* new_state, long long int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) {
        state[idx] = new_state[idx];
    }
}


__global__ void contract_tensor(
        const Complex* state,
        const Complex* gate,
        int qubit,
        Complex* new_state,
        const int* shape,
        int* new_idx,
        int* old_idx,
        const int n,
        const long long int N
        ) {
    extern __shared__ Complex shared_mem[]; // Use shared memory
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = idx * n;

    if (idx < N) {
        int temp = idx;

        // Compute the multi-dimensional index
        for (int i = n - 1; i >= 0; --i) {
            new_idx[offset + i] = temp % shape[i];
            temp /= shape[i];
        }

        // Copy new_idx to old_idx
        for (int i = 0; i < n; ++i) {
            old_idx[offset + i] = new_idx[offset + i];
        }

        // Compute the two values for j = 0 and j = 1 and store in shared memory
        for (int j = 0; j < 2; ++j) {
            old_idx[offset + qubit] = j;

            // Compute the linear index for old_idx
            int old_linear_idx = 0;
            int factor = 1;
            for (int i = n - 1; i >= 0; --i) {
                old_linear_idx += old_idx[offset + i] * factor;
                factor *= shape[i];
            }

            // Store the result in shared memory
            shared_mem[idx + j*N] = hipCmul(gate[new_idx[offset + qubit] * 2 + j], state[old_linear_idx]);
        }

        __syncthreads();

        // Perform reduction to sum two neighbor elements
        Complex val = shared_mem[idx];
        val.x += __shfl_down_sync(-1, val.x, 0);
        val.y += __shfl_down_sync(-1, val.y, 0);
        new_state[idx] = val;
    }
}


__global__ void applyPhaseFlip(Complex* state, long long int idx) {
    state[idx] = hipCmul(state[idx], make_hipDoubleComplex(-1.0, 0.0));
}


void applyGateAllQubits(
    Complex* state,
    const Complex* gate,
    Complex* new_state,
    const int* shape,
    int* new_idx,
    int* old_idx,
    int n,
    long long int N,
    dim3 dimBlock,
    dim3 dimGrid,
    int sharedMemSize
    ) {

    for (int i = 0; i < n; ++i) {
        contract_tensor<<<dimGrid, dimBlock, sharedMemSize>>>(state, gate, i, new_state, shape, new_idx, old_idx, n, N);
        // contract_tensor<<<dimGrid, dimBlock>>>(state, gate, i, new_state, shape, n, N);
        // cudaDeviceSynchronize();
        // Update the state with the new state
        updateState<<<dimGrid, dimBlock>>>(state, new_state, N);
        // cudaDeviceSynchronize();
        zeroOutState<<<dimGrid, dimBlock>>>(new_state, N);
        // cudaDeviceSynchronize();
    }
}

void applyGateSingleQubit(
    Complex* state,
    const Complex* gate,
    Complex* new_state,
    const int* shape,
    int* new_idx,
    int* old_idx,
    int n,
    long long int N,
    long long int idx,
    dim3 dimBlock,
    dim3 dimGrid,
    int sharedMemSize
    ) {

    contract_tensor<<<dimGrid, dimBlock, sharedMemSize>>>(state, gate, idx, new_state, shape, new_idx, old_idx, n, N);
    // Update the state with the new state
    updateState<<<dimGrid, dimBlock>>>(state, new_state, N);
    zeroOutState<<<dimGrid, dimBlock>>>(new_state, N);
}

void applyDiffusionOperator(
    Complex* state,
    Complex* new_state,
    const int* shape,
    const Complex* H,
    const Complex* X,
    const Complex* Z,
    int* new_idx,
    int* old_idx,
    int n,
    long long int N,
    dim3 dimBlock,
    dim3 dimGrid,
    int sharedMemSize
    ) {
    applyGateAllQubits(state, H, new_state, shape, new_idx, old_idx, n, N, dimBlock, dimGrid, sharedMemSize);
    applyGateAllQubits(state, X, new_state, shape, new_idx, old_idx, n, N, dimBlock, dimGrid, sharedMemSize);
    applyPhaseFlip<<<dimGrid, dimBlock>>>(state, N - 1);
    applyGateSingleQubit(state, Z, new_state, shape, new_idx, old_idx, n, N, 0, dimBlock, dimGrid, sharedMemSize);
    applyGateAllQubits(state, X, new_state, shape, new_idx, old_idx, n, N, dimBlock, dimGrid, sharedMemSize);
    applyGateSingleQubit(state, Z, new_state, shape, new_idx, old_idx, n, N, 0, dimBlock, dimGrid, sharedMemSize);
    applyGateAllQubits(state, H, new_state, shape, new_idx, old_idx, n, N, dimBlock, dimGrid, sharedMemSize);
}

// double* simulate(const Complex* weights, int numElements, int numSamples) {
//     if (numElements <= 0 || numSamples <= 0) {
//         fprintf(stderr, "Invalid input parameters.\n");
//         return NULL;
//     }

//     // Array to count occurrences of each index
//     int* counts = (int*)calloc(numElements, sizeof(int));
//     // Array to store the average frequencies
//     double* averages = (double*)calloc(numElements, sizeof(double));

//     if (counts == NULL || averages == NULL) {
//         fprintf(stderr, "Memory allocation failed.\n");
//         free(counts);
//         free(averages);
//         return NULL;
//     }

//     // Prepare weights for the distribution by extracting their magnitudes
//     double* magnitudes = (double*)malloc(numElements * sizeof(double));
//     if (magnitudes == NULL) {
//         fprintf(stderr, "Memory allocation failed.\n");
//         free(counts);
//         free(averages);
//         return NULL;
//     }

//     for (int i = 0; i < numElements; ++i) {
//         magnitudes[i] = cabs(weights[i]);
//     }

//     // Simulate the weighted distribution
//     for (int i = 0; i < numSamples; ++i) {
//         double r = (double)rand() / RAND_MAX;
//         double cum_prob = 0.0;
//         for (int j = 0; j < numElements; ++j) {
//             cum_prob += magnitudes[j];
//             if (r < cum_prob) {
//                 counts[j]++;
//                 break;
//             }
//         }
//     }

//     for (int i = 0; i < numElements; ++i) {
//         averages[i] = (double)counts[i] / numSamples;
//     }

//     free(counts);
//     free(magnitudes);
//     return averages;
// }

// Complex** createMatrix(int numRows, int numCols, const Complex* initialValues) {
//     if (numRows <= 0 || numCols <= 0) {
//         fprintf(stderr, "Invalid matrix dimensions.\n");
//         return NULL;
//     }

//     // Allocate memory for row pointers
//     Complex** matrix = (Complex**)malloc(numRows * sizeof(Complex*));
//     if (matrix == NULL) {
//         fprintf(stderr, "Memory allocation failed.\n");
//         return NULL;
//     }

//     // Allocate memory for each row and initialize with provided values
//     for (int i = 0; i < numRows; ++i) {
//         matrix[i] = (Complex*)malloc(numCols * sizeof(Complex));
//         if (matrix[i] == NULL) {
//             for (int j = 0; j < i; ++j) {
//                 free(matrix[j]);
//             }
//             free(matrix);
//             fprintf(stderr, "Memory allocation failed.\n");
//             return NULL;
//         }
//         for (int j = 0; j < numCols; ++j) {
//             int index = i * numCols + j;
//             matrix[i][j] = initialValues[index];
//         }
//     }

//     return matrix;
// }

// void deleteMatrix(Complex** matrix, int rows) {
//     for (int i = 0; i < rows; ++i) {
//         free(matrix[i]);
//     }
//     free(matrix);
// }

// Complex** kroneckerProduct(Complex** A, int aRows, int aCols, Complex** B, int bRows, int bCols) {
//     int resultRows = aRows * bRows;
//     int resultCols = aCols * bCols;
//     Complex** result = (Complex**)malloc(resultRows * sizeof(Complex*));
//     for (int i = 0; i < resultRows; ++i) {
//         result[i] = (Complex*)malloc(resultCols * sizeof(Complex));
//     }

//     for (int i = 0; i < aRows; ++i) {
//         for (int j = 0; j < aCols; ++j) {
//             for (int k = 0; k < bRows; ++k) {
//                 for (int l = 0; l < bCols;) {
//                     result[i * bRows + k][j * bCols + l] = A[i][j] * B[k][l];
//                 }
//             }
//         }
//     }

//     return result;
// }

// void printMatrix(Complex** matrix, int rows, int cols) {
//     for (int i = 0; i < rows; ++i) {
//         for (int j = 0; j < cols; ++j) {
//             printf("(%f + %fi) ", creal(matrix[i][j]), cimag(matrix[i][j]));
//         }
//         printf("\n");
//     }
// }
